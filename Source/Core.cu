#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"
#include "General.cuh"
#include "Framebuffer.cuh"
#include "Benchmark.cuh"

texture<unsigned short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexIntensity;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexGlossiness;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexEmission;

hipChannelFormatDesc gFloatChannelDesc = hipCreateChannelDesc<float>();
hipChannelFormatDesc gFloat4ChannelDesc = hipCreateChannelDesc<float4>();

hipArray* gpIntensity			= NULL;
hipArray* gpOpacity			= NULL;
hipArray* gpDiffuse			= NULL;
hipArray* gpSpecular			= NULL;
hipArray* gpGlossiness			= NULL;
hipArray* gpEmission			= NULL;

CD ErVolume			gVolume;
CD ErCamera			gCamera;
CD ErLights			gLights;
CD ErClippers		gClippers;
CD ErReflectors		gReflectors;
CD ErDenoise		gDenoise;
CD ErScattering		gScattering;
CD ErBlur			gBlur;
CD ErRange			gOpacityRange;
CD ErRange			gDiffuseRange;
CD ErRange			gSpecularRange;
CD ErRange			gGlossinessRange;
CD ErRange			gEmissionRange;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"
#include "GradientMagnitude.cuh"

FrameBuffer FB;

void ErInitialize()
{
}

void ErDeinitialize()
{
	ErUnbindDensityBuffer();
	ErUnbindOpacity1D();
	ErUnbindDiffuse1D();
	ErUnbindSpecular1D();
	ErUnbindGlossiness1D();
	ErUnbindEmission1D();
}

void ErResize(int Size[2])
{
	FB.Resize(CResolution2D(Size[0], Size[1]));
}

void ErResetFrameBuffer()
{
	FB.Reset();
}

void ErBindIntensityBuffer(unsigned short* pBuffer, int Extent[3])
{
	ErUnbindDensityBuffer();

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<unsigned short>();

	hipExtent CudaExtent = make_hipExtent(Extent[0], Extent[1], Extent[2]);

	HandleCudaError(hipMalloc3DArray(&gpIntensity, &ChannelDesc, CudaExtent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr		= make_hipPitchedPtr(pBuffer, CudaExtent.width * sizeof(unsigned short), CudaExtent.width, CudaExtent.height);
	CopyParams.dstArray		= gpIntensity;
	CopyParams.extent		= CudaExtent;
	CopyParams.kind			= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexIntensity.normalized		= true;
	gTexIntensity.filterMode		= hipFilterModeLinear;      
	gTexIntensity.addressMode[0]	= hipAddressModeClamp;  
	gTexIntensity.addressMode[1]	= hipAddressModeClamp;
  	gTexIntensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexIntensity, gpIntensity, ChannelDesc));
}

void ErUnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpIntensity));
	gpIntensity = NULL;
	HandleCudaError(hipUnbindTexture(gTexIntensity));
}

void ErBindOpacity1D(float Opacity[NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gOpacityRange"), &Int, sizeof(ErRange)));

	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	if (gpOpacity == NULL)
		HandleCudaError(hipMallocArray(&gpOpacity, &gFloatChannelDesc, NO_GRADIENT_STEPS, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacity, 0, 0, Opacity, NO_GRADIENT_STEPS * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacity, gFloatChannelDesc));
}

void ErBindDiffuse1D(float Diffuse[3][NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDiffuseRange"), &Int, sizeof(ErRange)));

	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	if (gpDiffuse == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuse, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1));

	ColorXYZAf* pDiffuseXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pDiffuseXYZA[i].FromRGB(Diffuse[0][i], Diffuse[1][i], Diffuse[2][i]);

	HandleCudaError(hipMemcpyToArray(gpDiffuse, 0, 0, pDiffuseXYZA, NO_GRADIENT_STEPS * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuse, gFloat4ChannelDesc));

	delete[] pDiffuseXYZA;
}

void ErBindSpecular1D(float Specular[3][NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gSpecularRange"), &Int, sizeof(ErRange)));

	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	if (gpSpecular == NULL)
		HandleCudaError(hipMallocArray(&gpSpecular, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1));

	ColorXYZAf* pSpecularXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pSpecularXYZA[i].FromRGB(Specular[0][i], Specular[1][i], Specular[2][i]);

	HandleCudaError(hipMemcpyToArray(gpSpecular, 0, 0, pSpecularXYZA, NO_GRADIENT_STEPS * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecular, gFloat4ChannelDesc));

	delete[] pSpecularXYZA;
}

void ErBindGlossiness1D(float Glossiness[NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gGlossinessRange"), &Int, sizeof(ErRange)));

	gTexGlossiness.normalized		= true;
	gTexGlossiness.filterMode		= hipFilterModeLinear;
	gTexGlossiness.addressMode[0]	= hipAddressModeClamp;

	if (gpGlossiness == NULL)
		HandleCudaError(hipMallocArray(&gpGlossiness, &gFloatChannelDesc, NO_GRADIENT_STEPS, 1));

	HandleCudaError(hipMemcpyToArray(gpGlossiness, 0, 0, Glossiness, NO_GRADIENT_STEPS * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexGlossiness, gpGlossiness, gFloatChannelDesc));
}

void ErBindEmission1D(float Emission[3][NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gEmissionRange"), &Int, sizeof(ErRange)));

	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	if (gpEmission == NULL)
		HandleCudaError(hipMallocArray(&gpEmission, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1));

	ColorXYZAf* pEmissionXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pEmissionXYZA[i].FromRGB(Emission[0][i], Emission[1][i], Emission[2][i]);

	HandleCudaError(hipMemcpyToArray(gpEmission, 0, 0, pEmissionXYZA, NO_GRADIENT_STEPS * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmission, gFloat4ChannelDesc));

	delete[] pEmissionXYZA;
}

void ErUnbindOpacity1D(void)
{
	HandleCudaError(hipFreeArray(gpOpacity));
	gpOpacity = NULL;
	HandleCudaError(hipUnbindTexture(gTexOpacity));
}

void ErUnbindDiffuse1D(void)
{
	HandleCudaError(hipFreeArray(gpDiffuse));
	gpDiffuse = NULL;
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
}

void ErUnbindSpecular1D(void)
{
	HandleCudaError(hipFreeArray(gpSpecular));
	gpSpecular	= NULL;
	HandleCudaError(hipUnbindTexture(gTexSpecular));
}

void ErUnbindGlossiness1D(void)
{
	HandleCudaError(hipFreeArray(gpGlossiness));
	gpGlossiness = NULL;
	HandleCudaError(hipUnbindTexture(gTexGlossiness));
}

void ErUnbindEmission1D(void)
{
	HandleCudaError(hipFreeArray(gpEmission));
	gpEmission	= NULL;
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void ErBindVolume(ErVolume* pVolume)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gVolume"), pVolume, sizeof(ErVolume)));
}

void ErBindCamera(ErCamera* pCamera)
{
	const Vec3f N = Normalize(ToVec3f(pCamera->m_Target) - ToVec3f(pCamera->m_Pos));
	const Vec3f U = Normalize(Cross(N, ToVec3f(pCamera->m_Up)));
	const Vec3f V = Normalize(Cross(N, U));

	pCamera->m_N[0] = N[0];
	pCamera->m_N[1] = N[1];
	pCamera->m_N[2] = N[2];

	pCamera->m_U[0] = U[0];
	pCamera->m_U[1] = U[1];
	pCamera->m_U[2] = U[2];

	pCamera->m_V[0] = V[0];
	pCamera->m_V[1] = V[1];
	pCamera->m_V[2] = V[2];

	float Scale = 0.0f;

	Scale = tanf((0.5f * pCamera->m_FOV / RAD_F));

	const float AspectRatio = (float)pCamera->m_FilmHeight / (float)pCamera->m_FilmWidth;

	if (AspectRatio > 1.0f)
	{
		pCamera->m_Screen[0][0] = -Scale;
		pCamera->m_Screen[0][1] = Scale;
		pCamera->m_Screen[1][0] = -Scale * AspectRatio;
		pCamera->m_Screen[1][1] = Scale * AspectRatio;
	}
	else
	{
		pCamera->m_Screen[0][0] = -Scale / AspectRatio;
		pCamera->m_Screen[0][1] = Scale / AspectRatio;
		pCamera->m_Screen[1][0] = -Scale;
		pCamera->m_Screen[1][1] = Scale;
	}

	pCamera->m_InvScreen[0] = (pCamera->m_Screen[0][1] - pCamera->m_Screen[0][0]) / (float)pCamera->m_FilmWidth;
	pCamera->m_InvScreen[1] = (pCamera->m_Screen[1][1] - pCamera->m_Screen[1][0]) / (float)pCamera->m_FilmHeight;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gCamera"), pCamera, sizeof(ErCamera)));
}

void ErBindLights(ErLights* pLights)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gLights"), pLights, sizeof(ErLights)));
}

void ErBindClippers(ErClippers* pClippers)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gClippers"), pClippers, sizeof(ErClippers)));
}

void ErBindReflectors(ErReflectors* pReflectors)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gReflectors"), pReflectors, sizeof(ErReflectors)));
}

void ErBindDenoise(ErDenoise* pDenoise)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoise"), pDenoise, sizeof(ErDenoise)));
}

void ErBindScattering(ErScattering* pScattering)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gScattering"), pScattering, sizeof(ErScattering)));
}

void ErBindBlur(ErBlur* pBlur)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gBlur"), pBlur, sizeof(ErBlur)));
}

void ErRenderEstimate()
{
	FrameBuffer* pDevFrameBuffer = NULL;
	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));
	HandleCudaError(hipMemcpy(pDevFrameBuffer, &FB, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	SingleScattering(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);
	BlurEstimate(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);
	ComputeEstimate(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);
	ToneMap(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);
//	ReduceNoise(pDevRenderInfo, pDevFrameBuffer, gCamera.m_FilmWidth, gCamera.m_FilmHeight);

	HandleCudaError(hipFree(pDevFrameBuffer));
}

void ErGetEstimate(unsigned char* pData)
{
	hipMemcpy(FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToHost);
	memcpy(pData, FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize());
}

void ErGetFrameEstimate(unsigned char* pData)
{
	//hipMemcpy(FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToHost);
	//memcpy(pData, FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize());
}

void ErGetDepthBuffer(unsigned char* pData)
{
	//hipMemcpy(FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToHost);
	//memcpy(pData, FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize());
}

void ErRecordBenchmarkImage()
{
	hipMemcpy(FB.BenchmarkEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToDevice);
}

void ErGetAverageNrmsError(float& AverageNrmsError)
{
	FrameBuffer* pDevFrameBuffer = NULL;
	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));
	HandleCudaError(hipMemcpy(pDevFrameBuffer, &FB, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	ComputeAverageNrmsError(FB, pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1], AverageNrmsError);

	HandleCudaError(hipFree(pDevFrameBuffer));
}

void ErGetMaximumGradientMagnitude(float& MaximumGradientMagnitude, int Extent[3])
{
	ComputeGradientMagnitudeVolume(Extent, MaximumGradientMagnitude);
}