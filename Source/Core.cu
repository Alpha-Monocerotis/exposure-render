/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#define __CUDA_ARCH__ 200

#include "ExposureRender.h"

#include "Tracer.cuh"

DEVICE ExposureRender::Tracer*			gpTracer			= NULL;
DEVICE ExposureRender::Volume* 			gpVolumes			= NULL;
DEVICE ExposureRender::Light*			gpLights			= NULL;
DEVICE ExposureRender::Object*			gpObjects			= NULL;
DEVICE ExposureRender::ClippingObject*	gpClippingObjects	= NULL;
DEVICE ExposureRender::Texture*			gpTextures			= NULL;

ExposureRender::CudaList<ExposureRender::Volume>			gVolumes("gpVolumes");
ExposureRender::CudaList<ExposureRender::Light>				gLights("gpLights");
ExposureRender::CudaList<ExposureRender::Object>			gObjects("gpObjects");
ExposureRender::CudaList<ExposureRender::ClippingObject>	gClippingObjects("gpClippingObjects");
ExposureRender::CudaList<ExposureRender::Texture>			gTextures("gpTextures");

#include "Utilities.cuh"

// Kernels
#include "SingleScattering.cuh"
#include "Estimate.cuh"
#include "ToneMap.cuh"

/*
#include "GaussianFilter.cuh"
#include "BilateralFilter.cuh"
#include "MedianFilter.cuh"

#include "GradientMagnitude.cuh"
#include "AutoFocus.cuh"
*/



#define EDIT_TRACER(id)												\
std::map<int, Tracer>::iterator	It;									\
It = gTracers.find(id);												\
if (It == gTracers.end())											\
	throw(Exception(Enums::Error, "Tracer does not exist!"));		\
Tracer& Tracer = gTracers[id];										

DEVICE ExposureRender::Tracer* pTracer = NULL;

void BindTracer(ExposureRender::Tracer& Tracer)
{
	if (pTracer == NULL)
		ExposureRender::CUDA::Allocate(pTracer);
	
	ExposureRender::CUDA::MemCopyHostToDevice(&Tracer, pTracer);
	ExposureRender::CUDA::MemCopyHostToDeviceSymbol(&pTracer, "gpTracer");
}

namespace ExposureRender
{

std::map<int, Tracer> gTracers;

EXPOSURE_RENDER_DLL void Resize(int TracerID, int Size[2])
{
	EDIT_TRACER(TracerID)

	Tracer.FrameBuffer.Resize(Resolution2i(Size[0], Size[1]));
}

EXPOSURE_RENDER_DLL void Reset(int TracerID)
{
	EDIT_TRACER(TracerID)

	Tracer.FrameBuffer.Reset();
	Tracer.NoIterations = 0;
}

EXPOSURE_RENDER_DLL void InitializeTracer(int& ID)
{
	if (ID < 0)
		ID = gTracers.size();

	gTracers[ID] = Tracer();

	gVolumes.Synchronize();
	gLights.Synchronize();
	gObjects.Synchronize();
	gClippingObjects.Synchronize();
}

EXPOSURE_RENDER_DLL void DeinitializeTracer(int ID)
{
	std::map<int, Tracer>::iterator	It;
	
	It = gTracers.find(ID);
	
	if (It != gTracers.end())
		gTracers.erase(ID);
}

EXPOSURE_RENDER_DLL void BindVolume(Volume V, int& ID)
{
	gVolumes.Bind(V, ID);
}

EXPOSURE_RENDER_DLL void UnbindVolume(int ID)
{
	gVolumes.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindLight(Light L, int& ID)
{
	gLights.Bind(L, ID);
}

EXPOSURE_RENDER_DLL void UnbindLight(int ID)
{
	gLights.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindObject(Object O, int& ID)
{
	gObjects.Bind(O, ID);
}

EXPOSURE_RENDER_DLL void UnbindObject(int ID)
{
	gObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindClippingObject(ClippingObject C, int& ID)
{
	gClippingObjects.Bind(C, ID);
}

EXPOSURE_RENDER_DLL void UnbindClippingObject(int ID)
{
	gClippingObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindTexture(Texture Texture, int& ID)
{
	gTextures.Bind(Texture, ID);
}

EXPOSURE_RENDER_DLL void UnbindTexture(int ID)
{
	gTextures.Unbind(ID);
}

EXPOSURE_RENDER_DLL void SetVolumeID(int TracerID, int VolumeID)
{
	EDIT_TRACER(TracerID)
	Tracer.VolumeID = VolumeID;
	Reset(TracerID);
}

EXPOSURE_RENDER_DLL void SetLightIDs(int TracerID, Indices LightIDs)
{
	EDIT_TRACER(TracerID)
	Tracer.BindLightIDs(LightIDs, gLights.HashMap);
	Reset(TracerID);
}

EXPOSURE_RENDER_DLL void SetObjectIDs(int TracerID, Indices ObjectIDs)
{
	EDIT_TRACER(TracerID)
	Tracer.BindObjectIDs(ObjectIDs, gObjects.HashMap);
	Reset(TracerID);
}

EXPOSURE_RENDER_DLL void SetClippingObjectIDs(int TracerID, Indices ClippingObjectIDs)
{
	EDIT_TRACER(TracerID)
	Tracer.BindClippingObjectIDs(ClippingObjectIDs, gClippingObjects.HashMap);
	Reset(TracerID);
}

EXPOSURE_RENDER_DLL void BindOpacity1D(int TracerID, ScalarTransferFunction1D Opacity1D)
{
	EDIT_TRACER(TracerID)
	Tracer.Opacity1D = Opacity1D;
}

EXPOSURE_RENDER_DLL void BindDiffuse1D(int TracerID, ColorTransferFunction1D Diffuse1D)
{
	EDIT_TRACER(TracerID)
	Tracer.Diffuse1D = Diffuse1D;
}

EXPOSURE_RENDER_DLL void BindSpecular1D(int TracerID, ColorTransferFunction1D Specular1D)
{
	EDIT_TRACER(TracerID)
	Tracer.Specular1D = Specular1D;
}

EXPOSURE_RENDER_DLL void BindGlossiness1D(int TracerID, ScalarTransferFunction1D Glossiness1D)
{
	EDIT_TRACER(TracerID)
	Tracer.Glossiness1D = Glossiness1D;
}

EXPOSURE_RENDER_DLL void BindEmission1D(int TracerID, ColorTransferFunction1D Emission1D)
{
	EDIT_TRACER(TracerID)
	Tracer.Emission1D = Emission1D;
}

EXPOSURE_RENDER_DLL void BindCamera(int TracerID, Camera Camera)
{
	EDIT_TRACER(TracerID)
	Tracer.Camera = Camera;
	Tracer.Camera.ToDevice();
}

EXPOSURE_RENDER_DLL void BindRenderSettings(int TracerID, RenderSettings RenderSettings)
{
	EDIT_TRACER(TracerID)
	Tracer.RenderSettings = RenderSettings;
}

EXPOSURE_RENDER_DLL void RenderEstimate(int TracerID)
{
	EDIT_TRACER(TracerID)

	BindTracer(Tracer);

	SingleScattering(Tracer.FrameBuffer.Resolution[0], Tracer.FrameBuffer.Resolution[1]);
	ComputeEstimate(Tracer.FrameBuffer.Resolution[0], Tracer.FrameBuffer.Resolution[1]);
	ToneMap(Tracer.FrameBuffer.Resolution[0], Tracer.FrameBuffer.Resolution[1]);

	Tracer.NoIterations++; 
}

EXPOSURE_RENDER_DLL void GetEstimate(int TracerID, unsigned char* pData)
{
	EDIT_TRACER(TracerID)
	CUDA::MemCopyDeviceToHost(Tracer.FrameBuffer.CudaDisplayEstimate.GetPtr(), (ColorRGBAuc*)pData, Tracer.FrameBuffer.CudaDisplayEstimate.GetNoElements());
}

EXPOSURE_RENDER_DLL void GetAutoFocusDistance(int TracerID, int FilmU, int FilmV, float& AutoFocusDistance)
{
	return;
//	ComputeAutoFocusDistance(FilmU, FilmV, AutoFocusDistance);
}

EXPOSURE_RENDER_DLL void GetNoIterations(int TracerID, int& NoIterations)
{
	EDIT_TRACER(TracerID)
	NoIterations = Tracer.NoIterations; 
}

}
