/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "General.cuh"
#include "Defines.cuh"
#include "Core.cuh"

DEVICE int* gpTracer = NULL;

#include "Volume.cuh"
#include "Light.cuh"
#include "Object.cuh"
#include "ClippingObject.cuh"
#include "Texture.cuh"

DEVICE Volumes* gpVolumes					= NULL;
DEVICE Objects* gpObjects					= NULL;
DEVICE ClippingObjects*	gpClippingObjects	= NULL;
DEVICE Textures* gpTextures					= NULL;

#include "Shared.cuh"
#include "Tracer.cuh"
#include "Utilities.cuh"

#include "GaussianFilter.cuh"
#include "BilateralFilter.cuh"
#include "MedianFilter.cuh"
#include "Estimate.cuh"

#include "SingleScattering.cuh"
#include "ToneMap.cuh"
#include "GradientMagnitude.cuh"
#include "AutoFocus.cuh"

namespace ExposureRender
{

std::map<int, Tracer> gTracers;

SharedResources<Volume, MAX_NO_VOLUMES>						gSharedVolumes("gpVolumes");
SharedResources<Object, MAX_NO_OBJECTS>						gSharedObjects("gpObjects");
SharedResources<Light, MAX_NO_LIGHTS>						gSharedLights("gpLights");
SharedResources<ClippingObject, MAX_NO_CLIPPING_OBJECTS>	gSharedClippingObjects("gpClippingObjects");
SharedResources<Texture, MAX_NO_TEXTURES>					gSharedTextures("gpTextures");

EXPOSURE_RENDER_DLL void Resize(int TracerID, int Size[2])
{
	gTracers[TracerID].FrameBuffer.Resize(Resolution2i(Size));
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void Reset(int TracerID)
{
	gTracers[TracerID].FrameBuffer.Reset();
	gTracers[TracerID].NoIterations = 0;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void InitializeTracer(int& ID)
{
//	hipSetDevice(0);

//	gTracers.Bind(Tracer(), ID);
}

EXPOSURE_RENDER_DLL void DeinitializeTracer(int ID)
{
//	gTracers.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindVolume(ErVolume V, int& ID)
{
//	gSharedVolumes.Bind(Volume(V), ID);
}

EXPOSURE_RENDER_DLL void UnbindVolume(int ID)
{
//	gSharedVolumes.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindLight(ErLight L, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindLight(int ID)
{
//	gSharedLights.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindObject(ErObject O, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindObject(int ID)
{
//	gSharedObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindClippingObject(ErClippingObject C, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindClippingObject(int ID)
{
//	gSharedClippingObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindTexture(ErTexture Texture, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindTexture(int ID)
{
//	gSharedTextures.Unbind(ID);
}

EXPOSURE_RENDER_DLL void SetTracerVolumeIDs(int ID[MAX_NO_VOLUMES], int Size)
{
}

EXPOSURE_RENDER_DLL void SetTracerLightIDs(int ID[MAX_NO_LIGHTS], int Size)
{
}

EXPOSURE_RENDER_DLL void SetTracerObjectIDs(int ID[MAX_NO_OBJECTS], int Size)
{
}

EXPOSURE_RENDER_DLL void SetTracerClippingObjectIDs(int ID[MAX_NO_CLIPPING_OBJECTS], int Size)
{
}

EXPOSURE_RENDER_DLL void BindOpacity1D(int TracerID, ErScalarTransferFunction1D Opacity1D)
{
	gTracers[TracerID].Opacity1D = Opacity1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindDiffuse1D(int TracerID, ErColorTransferFunction1D Diffuse1D)
{
	gTracers[TracerID].Diffuse1D = Diffuse1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindSpecular1D(int TracerID, ErColorTransferFunction1D Specular1D)
{
	gTracers[TracerID].Specular1D = Specular1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindGlossiness1D(int TracerID, ErScalarTransferFunction1D Glossiness1D)
{
	gTracers[TracerID].Glossiness1D = Glossiness1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindEmission1D(int TracerID, ErColorTransferFunction1D Emission1D)
{
	gTracers[TracerID].Emission1D = Emission1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindCamera(int TracerID, ErCamera Camera)
{
	gTracers[TracerID].Camera = Camera;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindRenderSettings(int TracerID, ErRenderSettings RenderSettings)
{
	gTracers[TracerID].RenderSettings = RenderSettings;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindFiltering(int TracerID, ErFiltering Filtering)
{
}

EXPOSURE_RENDER_DLL void RenderEstimate(int TracerID)
{
//	CUDA::HostToConstantDevice(&TracerID, gActiveTracerID);

	CUDA::ThreadSynchronize();

//	SingleScattering(gTracers[TracerID].FrameBuffer.Resolution[0], gTracers[TracerID].FrameBuffer.Resolution[1]);
//	ComputeEstimate(gTracers[TracerID].FrameBuffer.Resolution[0], gTracers[TracerID].FrameBuffer.Resolution[1]);
//	ToneMap(gTracers[TracerID].FrameBuffer.Resolution[0], gTracers[TracerID].FrameBuffer.Resolution[1]);

	CUDA::ThreadSynchronize();

	gTracers[TracerID].NoIterations++; 
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void GetEstimate(int TracerID, unsigned char* pData)
{
	CUDA::MemCopyDeviceToHost(gTracers[TracerID].FrameBuffer.CudaDisplayEstimate.GetPtr(), (ColorRGBAuc*)pData, gTracers[TracerID].FrameBuffer.CudaDisplayEstimate.GetNoElements());
}

EXPOSURE_RENDER_DLL void GetAutoFocusDistance(int TracerID, int FilmU, int FilmV, float& AutoFocusDistance)
{
	return;
	ComputeAutoFocusDistance(FilmU, FilmV, AutoFocusDistance);
}

EXPOSURE_RENDER_DLL void GetNoIterations(int TracerID, int& NoIterations)
{
	NoIterations = gTracers[TracerID].NoIterations; 
}

}
