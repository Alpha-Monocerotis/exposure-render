#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"

#include "Slice.cuh"

#include "VolumeInfo.cuh"
#include "RenderInfo.cuh"

texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, hipTextureType3D, hipReadModeElementType>			gTexExtinction;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexRoughness;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
hipArray* gpOpacityArray				= NULL;
hipArray* gpDiffuseArray				= NULL;
hipArray* gpSpecularArray				= NULL;
hipArray* gpRoughnessArray				= NULL;
hipArray* gpEmissionArray				= NULL;

CD VolumeInfo	gVolumeInfo;

#define TF_NO_SAMPLES		128
#define INV_TF_NO_SAMPLES	1.0f / (float)TF_NO_SAMPLES

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

void BindIntensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();

	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexGradientMagnitude));
}

void BindTransferFunctionOpacity(CTransferFunction& TransferFunctionOpacity)
{
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	float Opacity[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Opacity[i] = TransferFunctionOpacity.F((float)i * INV_TF_NO_SAMPLES)[1];
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, TF_NO_SAMPLES * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, ChannelDesc));
}

void UnbindTransferFunctionOpacity(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	gpOpacityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexOpacity));
}

void BindTransferFunctionDiffuse(CTransferFunction& TransferFunctionDiffuse)
{
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	float4 Diffuse[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		ColorXYZAf Color;
		
		ColorRGBf ColorRgbHdr = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES);
		
		Color.FromRGB(ColorRgbHdr[0], ColorRgbHdr[1], ColorRgbHdr[2]);

		Diffuse[i].x = Color.GetX();
		Diffuse[i].y = Color.GetY();
		Diffuse[i].z = Color.GetZ();
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, Diffuse, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, ChannelDesc));
}

void UnbindTransferFunctionDiffuse(void)
{
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	gpDiffuseArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
}

void BindTransferFunctionSpecular(CTransferFunction& TransferFunctionSpecular)
{
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	float4 Specular[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		ColorXYZAf Color;
		
		ColorRGBf ColorRgbHdr = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES);
		
		Color.FromRGB(ColorRgbHdr[0], ColorRgbHdr[1], ColorRgbHdr[2]);

		Specular[i].x = Color.GetX();
		Specular[i].y = Color.GetY();
		Specular[i].z = Color.GetZ();
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, Specular, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, ChannelDesc));
}

void UnbindTransferFunctionSpecular(void)
{
	HandleCudaError(hipFreeArray(gpSpecularArray));
	gpSpecularArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexSpecular));
}

void BindTransferFunctionRoughness(CTransferFunction& TransferFunctionRoughness)
{
	gTexRoughness.normalized		= true;
	gTexRoughness.filterMode		= hipFilterModeLinear;
	gTexRoughness.addressMode[0]	= hipAddressModeClamp;

	float Roughness[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Roughness[i] = TransferFunctionRoughness.F((float)i * INV_TF_NO_SAMPLES)[0];
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpRoughnessArray == NULL)
		HandleCudaError(hipMallocArray(&gpRoughnessArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpRoughnessArray, 0, 0, Roughness, TF_NO_SAMPLES * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexRoughness, gpRoughnessArray, ChannelDesc));
}

void UnbindTransferFunctionRoughness(void)
{
	HandleCudaError(hipFreeArray(gpRoughnessArray));
	gpRoughnessArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexRoughness));
}

void BindTransferFunctionEmission(CTransferFunction& TransferFunctionEmission)
{
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	float4 Emission[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		ColorXYZAf Color;
		
		ColorRGBf ColorRgbHdr = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES);
		
		Color.FromRGB(ColorRgbHdr[0], ColorRgbHdr[1], ColorRgbHdr[2]);

		Emission[i].x = Color.GetX();
		Emission[i].y = Color.GetY();
		Emission[i].z = Color.GetZ();
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, Emission, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, ChannelDesc));
}

void UnbindTransferFunctionEmission(void)
{
	HandleCudaError(hipFreeArray(gpEmissionArray));
	gpEmissionArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void RenderEstimate(VolumeInfo* pVolumeInfo, RenderInfo* pRenderInfo, FrameBuffer* pFrameBuffer)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gVolumeInfo"), pVolumeInfo, sizeof(VolumeInfo)));

//	VolumeInfo*		pDevVolumeInfo	= NULL;
	RenderInfo*		pDevRenderInfo	= NULL;
	FrameBuffer*	pDevFrameBuffer	= NULL;

//	HandleCudaError(hipMalloc(&pDevVolumeInfo, sizeof(VolumeInfo)));
	HandleCudaError(hipMalloc(&pDevRenderInfo, sizeof(RenderInfo)));
	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));

//	HandleCudaError(hipMemcpy(pDevVolumeInfo, pVolumeInfo, sizeof(VolumeInfo), hipMemcpyHostToDevice));
	HandleCudaError(hipMemcpy(pDevRenderInfo, pRenderInfo, sizeof(RenderInfo), hipMemcpyHostToDevice));
	HandleCudaError(hipMemcpy(pDevFrameBuffer, pFrameBuffer, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	const dim3 BlockDim(8, 8);
	const dim3 GridDim((int)ceilf((float)pRenderInfo->m_FilmWidth / (float)BlockDim.x), (int)ceilf((float)pRenderInfo->m_FilmHeight / (float)BlockDim.y));

	SingleScattering(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
//	Blur(BlockDim, GridDim, pDevRenderInfo);
//	Estimate(BlockDim, GridDim, pDevRenderInfo);
//	ToneMap(BlockDim, GridDim, pDevRenderInfo);
//	Denoise(BlockDim, GridDim, pDevRenderInfo);

//	HandleCudaError(hipFree(pDevVolumeInfo));
	HandleCudaError(hipFree(pDevRenderInfo));
	HandleCudaError(hipFree(pDevFrameBuffer));
}