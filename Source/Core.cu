#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"
#include "General.cuh"
#include "Buffer.cuh"

texture<unsigned short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexIntensity;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexEnvironmentGradient;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexGlossiness;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexIor;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexEmission;

hipChannelFormatDesc gFloatChannelDesc = hipCreateChannelDesc<float>();
hipChannelFormatDesc gFloat4ChannelDesc = hipCreateChannelDesc<float4>();

hipArray* gpIntensity			= NULL;
hipArray* gEnvironmentGradient	= NULL;
hipArray* gpOpacity			= NULL;
hipArray* gpDiffuse			= NULL;
hipArray* gpSpecular			= NULL;
hipArray* gpGlossiness			= NULL;
hipArray* gpIor				= NULL;
hipArray* gpEmission			= NULL;

CD _Volume			gVolume;
CD _Camera			gCamera;
CD _Lighting		gLighting;
CD _Clipping		gClipping;
CD _Reflectors		gReflectors;
CD _Denoise			gDenoise;
CD _Scattering		gScattering;
CD _Blur			gBlur;

CD Interval			gOpacityRange;
CD Interval			gDiffuseRange;
CD Interval			gSpecularRange;
CD Interval			gGlossinessRange;
CD Interval			gIorRange;
CD Interval			gEmissionRange;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

FrameBuffer FB;

void ErInitialize()
{
}

void ErDeinitialize()
{
}

void ErResize(int Size[2])
{
	FB.Resize(CResolution2D(Size[0], Size[1]));
}

void ErResetFrameBuffer()
{
	FB.Reset();
}

void ErBindIntensityBuffer(unsigned short* pBuffer, int Extent[3])
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<unsigned short>();

	hipExtent CudaExtent = make_hipExtent(Extent[0], Extent[1], Extent[2]);

	HandleCudaError(hipMalloc3DArray(&gpIntensity, &ChannelDesc, CudaExtent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr		= make_hipPitchedPtr(pBuffer, CudaExtent.width * sizeof(unsigned short), CudaExtent.width, CudaExtent.height);
	CopyParams.dstArray		= gpIntensity;
	CopyParams.extent		= CudaExtent;
	CopyParams.kind			= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexIntensity.normalized		= true;
	gTexIntensity.filterMode		= hipFilterModeLinear;      
	gTexIntensity.addressMode[0]	= hipAddressModeClamp;  
	gTexIntensity.addressMode[1]	= hipAddressModeClamp;
  	gTexIntensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexIntensity, gpIntensity, ChannelDesc));
}

void ErUnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpIntensity));
	gpIntensity = NULL;
	HandleCudaError(hipUnbindTexture(gTexIntensity));
}

void ErBindEnvironmentGradient(float EnvironmentGradient[3][NO_GRADIENT_STEPS])
{
	gTexEnvironmentGradient.normalized		= true;
	gTexEnvironmentGradient.filterMode		= hipFilterModeLinear;
	gTexEnvironmentGradient.addressMode[0]	= hipAddressModeClamp;

	if (gEnvironmentGradient == NULL)
		HandleCudaError(hipMallocArray(&gEnvironmentGradient, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1));

	ColorXYZAf* pEnvironmentGradientXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pEnvironmentGradientXYZA[i].FromRGB(EnvironmentGradient[0][i], EnvironmentGradient[1][i], EnvironmentGradient[2][i]);

	HandleCudaError(hipMemcpyToArray(gEnvironmentGradient, 0, 0, pEnvironmentGradientXYZA, NO_GRADIENT_STEPS * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEnvironmentGradient, gEnvironmentGradient, gFloat4ChannelDesc));

	delete[] pEnvironmentGradientXYZA;
}

void ErUnbindEnvironmentGradient(void)
{
	HandleCudaError(hipFreeArray(gEnvironmentGradient));
	gEnvironmentGradient = NULL;
	HandleCudaError(hipUnbindTexture(gTexEnvironmentGradient));
}

void ErBindOpacity1D(float Opacity[NO_GRADIENT_STEPS], float Range[2])
{
	Interval Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gOpacityRange"), &Int, sizeof(Interval)));

	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	if (gpOpacity == NULL)
		HandleCudaError(hipMallocArray(&gpOpacity, &gFloatChannelDesc, NO_GRADIENT_STEPS, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacity, 0, 0, Opacity, NO_GRADIENT_STEPS * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacity, gFloatChannelDesc));
}

void ErBindDiffuse1D(float Diffuse[3][NO_GRADIENT_STEPS], float Range[2])
{
	Interval Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDiffuseRange"), &Int, sizeof(Interval)));

	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	if (gpDiffuse == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuse, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1));

	ColorXYZAf* pDiffuseXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pDiffuseXYZA[i].FromRGB(Diffuse[0][i], Diffuse[1][i], Diffuse[2][i]);

	HandleCudaError(hipMemcpyToArray(gpDiffuse, 0, 0, pDiffuseXYZA, NO_GRADIENT_STEPS * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuse, gFloat4ChannelDesc));

	delete[] pDiffuseXYZA;
}

void ErBindSpecular1D(float Specular[3][NO_GRADIENT_STEPS], float Range[2])
{
	Interval Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gSpecularRange"), &Int, sizeof(Interval)));

	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	if (gpSpecular == NULL)
		HandleCudaError(hipMallocArray(&gpSpecular, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1));

	ColorXYZAf* pSpecularXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pSpecularXYZA[i].FromRGB(Specular[0][i], Specular[1][i], Specular[2][i]);

	HandleCudaError(hipMemcpyToArray(gpSpecular, 0, 0, pSpecularXYZA, NO_GRADIENT_STEPS * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecular, gFloat4ChannelDesc));

	delete[] pSpecularXYZA;
}

void ErBindGlossiness1D(float Glossiness[NO_GRADIENT_STEPS], float Range[2])
{
	Interval Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gGlossinessRange"), &Int, sizeof(Interval)));

	gTexGlossiness.normalized		= true;
	gTexGlossiness.filterMode		= hipFilterModeLinear;
	gTexGlossiness.addressMode[0]	= hipAddressModeClamp;

	if (gpGlossiness == NULL)
		HandleCudaError(hipMallocArray(&gpGlossiness, &gFloatChannelDesc, NO_GRADIENT_STEPS, 1));

	HandleCudaError(hipMemcpyToArray(gpGlossiness, 0, 0, Glossiness, NO_GRADIENT_STEPS * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexGlossiness, gpGlossiness, gFloatChannelDesc));
}

void ErBindIor1D(float Ior[NO_GRADIENT_STEPS], float Range[2])
{
	Interval Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gIorRange"), &Int, sizeof(Interval)));

	gTexIor.normalized		= true;
	gTexIor.filterMode		= hipFilterModeLinear;
	gTexIor.addressMode[0]	= hipAddressModeClamp;

	if (gpIor == NULL)
		HandleCudaError(hipMallocArray(&gpIor, &gFloatChannelDesc, NO_GRADIENT_STEPS, 1));

	HandleCudaError(hipMemcpyToArray(gpIor, 0, 0, Ior, NO_GRADIENT_STEPS * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexIor, gpIor, gFloatChannelDesc));
}

void ErBindEmission1D(float Emission[3][NO_GRADIENT_STEPS], float Range[2])
{
	Interval Int;
	Int.Set(Range);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gEmissionRange"), &Int, sizeof(Interval)));

	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	if (gpEmission == NULL)
		HandleCudaError(hipMallocArray(&gpEmission, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1));

	ColorXYZAf* pEmissionXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pEmissionXYZA[i].FromRGB(Emission[0][i], Emission[1][i], Emission[2][i]);

	HandleCudaError(hipMemcpyToArray(gpEmission, 0, 0, pEmissionXYZA, NO_GRADIENT_STEPS * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmission, gFloat4ChannelDesc));

	delete[] pEmissionXYZA;
}

void ErUnbindOpacity1D(void)
{
	HandleCudaError(hipFreeArray(gpOpacity));
	gpOpacity = NULL;
	HandleCudaError(hipUnbindTexture(gTexOpacity));
}

void ErUnbindDiffuse1D(void)
{
	HandleCudaError(hipFreeArray(gpDiffuse));
	gpDiffuse = NULL;
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
}

void ErUnbindSpecular1D(void)
{
	HandleCudaError(hipFreeArray(gpSpecular));
	gpSpecular	= NULL;
	HandleCudaError(hipUnbindTexture(gTexSpecular));
}

void ErUnbindGlossiness1D(void)
{
	HandleCudaError(hipFreeArray(gpGlossiness));
	gpGlossiness = NULL;
	HandleCudaError(hipUnbindTexture(gTexGlossiness));
}

void ErUnbindIor1D(void)
{
	HandleCudaError(hipFreeArray(gpIor));
	gpIor = NULL;
	HandleCudaError(hipUnbindTexture(gTexIor));
}

void ErUnbindEmission1D(void)
{
	HandleCudaError(hipFreeArray(gpEmission));
	gpEmission	= NULL;
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void ErBindVolume(_Volume* pVolume)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gVolume"), pVolume, sizeof(_Volume)));
}

void ErBindCamera(_Camera* pCamera)
{
	const Vec3f N = Normalize(ToVec3f(pCamera->m_Target) - ToVec3f(pCamera->m_Pos));
	const Vec3f U = Normalize(Cross(N, ToVec3f(pCamera->m_Up)));
	const Vec3f V = Normalize(Cross(N, U));

	pCamera->m_N[0] = N[0];
	pCamera->m_N[1] = N[1];
	pCamera->m_N[2] = N[2];

	pCamera->m_U[0] = U[0];
	pCamera->m_U[1] = U[1];
	pCamera->m_U[2] = U[2];

	pCamera->m_V[0] = V[0];
	pCamera->m_V[1] = V[1];
	pCamera->m_V[2] = V[2];

	float Scale = 0.0f;

	Scale = tanf((0.5f * pCamera->m_FOV / RAD_F));

	const float AspectRatio = (float)pCamera->m_FilmHeight / (float)pCamera->m_FilmWidth;

	if (AspectRatio > 1.0f)
	{
		pCamera->m_Screen[0][0] = -Scale;
		pCamera->m_Screen[0][1] = Scale;
		pCamera->m_Screen[1][0] = -Scale * AspectRatio;
		pCamera->m_Screen[1][1] = Scale * AspectRatio;
	}
	else
	{
		pCamera->m_Screen[0][0] = -Scale / AspectRatio;
		pCamera->m_Screen[0][1] = Scale / AspectRatio;
		pCamera->m_Screen[1][0] = -Scale;
		pCamera->m_Screen[1][1] = Scale;
	}

	pCamera->m_InvScreen[0] = (pCamera->m_Screen[0][1] - pCamera->m_Screen[0][0]) / (float)pCamera->m_FilmWidth;
	pCamera->m_InvScreen[1] = (pCamera->m_Screen[1][1] - pCamera->m_Screen[1][0]) / (float)pCamera->m_FilmHeight;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gCamera"), pCamera, sizeof(_Camera)));
}

void ErBindLighting(_Lighting* pLighting)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gLighting"), pLighting, sizeof(_Lighting)));
}

void ErBindClipping(_Clipping* pClipping)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gClipping"), pClipping, sizeof(_Clipping)));
}

void ErBindReflectors(_Reflectors* pReflectors)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gReflectors"), pReflectors, sizeof(_Reflectors)));
}

void ErBindDenoise(_Denoise* pDenoise)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoise"), pDenoise, sizeof(_Denoise)));
}

void ErBindScattering(_Scattering* pScattering)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gScattering"), pScattering, sizeof(_Scattering)));
}

void ErBindBlur(_Blur* pBlur)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gBlur"), pBlur, sizeof(_Blur)));
}

void ErRenderEstimate()
{
	FrameBuffer* pDevFrameBuffer = NULL;

	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));

//	HandleCudaError(hipMemcpy(pDevFrameBuffer, pFrameBuffer, sizeof(FrameBuffer), hipMemcpyHostToDevice));
	HandleCudaError(hipMemcpy(pDevFrameBuffer, &FB, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	SingleScattering(pDevFrameBuffer, FB.m_Resolution[0], FB.m_Resolution[1]);
	BlurEstimate(pDevFrameBuffer, FB.m_Resolution[0], FB.m_Resolution[1]);
	ComputeEstimate(pDevFrameBuffer, FB.m_Resolution[0], FB.m_Resolution[1]);
	ToneMap(pDevFrameBuffer, FB.m_Resolution[0], FB.m_Resolution[1]);
//	ReduceNoise(pDevRenderInfo, pDevFrameBuffer, gCamera.m_FilmWidth, gCamera.m_FilmHeight);

	HandleCudaError(hipFree(pDevFrameBuffer));
}

void ErGetRenderBuffer(unsigned char* pData)
{
	hipMemcpy(FB.m_DisplayEstimateRgbaLdrHost.GetPtr(), FB.m_EstimateRgbaLdr.GetPtr(), FB.m_EstimateRgbaLdr.GetSize(), hipMemcpyDeviceToHost);
	memcpy(pData, FB.m_DisplayEstimateRgbaLdrHost.GetPtr(), FB.m_EstimateRgbaLdr.GetSize());
}