#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"

#include "General.cuh"

texture<float, hipTextureType3D, hipReadModeElementType>			gTexIntensity;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexGlossiness;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexIOR;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipChannelFormatDesc gFloatChannelDesc = hipCreateChannelDesc<float>();
hipChannelFormatDesc gFloat4ChannelDesc = hipCreateChannelDesc<float4>();

hipArray* gpIntensity			= NULL;
hipArray* gpExtinction			= NULL;
hipArray* gpGradientMagnitude	= NULL;
hipArray* gpOpacity			= NULL;
hipArray* gpDiffuse			= NULL;
hipArray* gpSpecular			= NULL;
hipArray* gpGlossiness			= NULL;
hipArray* gpIOR				= NULL;
hipArray* gpEmission			= NULL;

CD Volume		gVolume;
CD Camera		gCamera;
CD Lighting		gLighting;
CD Slicing		gSlicing;
CD Denoise		gDenoise;
CD Scattering	gScattering;
CD Blur			gBlur;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

void BindIntensityBuffer(float* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	HandleCudaError(hipMalloc3DArray(&gpIntensity, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(float), Extent.width, Extent.height);
	CopyParams.dstArray	= gpIntensity;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexIntensity.normalized		= true;
	gTexIntensity.filterMode		= hipFilterModeLinear;      
	gTexIntensity.addressMode[0]	= hipAddressModeClamp;  
	gTexIntensity.addressMode[1]	= hipAddressModeClamp;
  	gTexIntensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexIntensity, gpIntensity, ChannelDesc));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpIntensity));
	gpIntensity = NULL;
	HandleCudaError(hipUnbindTexture(gTexIntensity));
}

void BindTransferFunctions1D(float Opacity[128], float Diffuse[3][128], float Specular[3][128], float Glossiness[128], float IOR[128], float Emission[3][128], int N)
{
	// Opacity
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	if (gpOpacity == NULL)
		HandleCudaError(hipMallocArray(&gpOpacity, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacity, 0, 0, Opacity, N * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacity, gFloatChannelDesc));

	// Diffuse
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	if (gpDiffuse == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuse, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pDiffuseXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pDiffuseXYZA[i].FromRGB(Diffuse[0][i], Diffuse[1][i], Diffuse[2][i]);

	HandleCudaError(hipMemcpyToArray(gpDiffuse, 0, 0, pDiffuseXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuse, gFloat4ChannelDesc));

	delete[] pDiffuseXYZA;

	// Specular
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	if (gpSpecular == NULL)
		HandleCudaError(hipMallocArray(&gpSpecular, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pSpecularXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pSpecularXYZA[i].FromRGB(Specular[0][i], Specular[1][i], Specular[2][i]);

	HandleCudaError(hipMemcpyToArray(gpSpecular, 0, 0, pSpecularXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecular, gFloat4ChannelDesc));

	delete[] pSpecularXYZA;

	// Glossiness
	gTexGlossiness.normalized		= true;
	gTexGlossiness.filterMode		= hipFilterModeLinear;
	gTexGlossiness.addressMode[0]	= hipAddressModeClamp;

	if (gpGlossiness == NULL)
		HandleCudaError(hipMallocArray(&gpGlossiness, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpGlossiness, 0, 0, Glossiness, N * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexGlossiness, gpGlossiness, gFloatChannelDesc));

	// IOR
	gTexIOR.normalized		= true;
	gTexIOR.filterMode		= hipFilterModeLinear;
	gTexIOR.addressMode[0]	= hipAddressModeClamp;

	if (gpIOR == NULL)
		HandleCudaError(hipMallocArray(&gpIOR, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpIOR, 0, 0, IOR, N * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexIOR, gpIOR, gFloatChannelDesc));

	// Emission
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	if (gpEmission == NULL)
		HandleCudaError(hipMallocArray(&gpEmission, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pEmissionXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pEmissionXYZA[i].FromRGB(Emission[0][i], Emission[1][i], Emission[2][i]);

	HandleCudaError(hipMemcpyToArray(gpEmission, 0, 0, pEmissionXYZA, N * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmission, gFloat4ChannelDesc));

	delete[] pEmissionXYZA;
}

void UnbindTransferFunctions1D(void)
{
	HandleCudaError(hipFreeArray(gpOpacity));
	HandleCudaError(hipFreeArray(gpDiffuse));
	HandleCudaError(hipFreeArray(gpSpecular));
	HandleCudaError(hipFreeArray(gpGlossiness));
	HandleCudaError(hipFreeArray(gpEmission));

	gpOpacity		= NULL;
	gpDiffuse		= NULL;
	gpSpecular		= NULL;
	gpGlossiness	= NULL;
	gpEmission		= NULL;

	HandleCudaError(hipUnbindTexture(gTexOpacity));
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
	HandleCudaError(hipUnbindTexture(gTexSpecular));
	HandleCudaError(hipUnbindTexture(gTexGlossiness));
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void RenderEstimate(Volume* pVolume, Camera* pCamera, Lighting* pLighting, Slicing* pSlicing, Denoise* pDenoise, Scattering* pScattering, Blur* pBlur, FrameBuffer* pFrameBuffer)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gVolume"), pVolume, sizeof(Volume)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gCamera"), pCamera, sizeof(Camera)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gLighting"), pLighting, sizeof(Lighting)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gSlicing"), pSlicing, sizeof(Slicing)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoise"), pDenoise, sizeof(Denoise)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gScattering"), pScattering, sizeof(Scattering)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gBlur"), pBlur, sizeof(Blur)));
	
	FrameBuffer* pDevFrameBuffer = NULL;

	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));

	HandleCudaError(hipMemcpy(pDevFrameBuffer, pFrameBuffer, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	const dim3 BlockDim(8, 8);
	const dim3 GridDim((int)ceilf((float)gCamera.m_FilmWidth / (float)BlockDim.x), (int)ceilf((float)gCamera.m_FilmHeight / (float)BlockDim.y));

	SingleScattering(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
	BlurEstimate(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
	ComputeEstimate(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
	ToneMap(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
//	ReduceNoise(pDevRenderInfo, pDevFrameBuffer, gCamera.m_FilmWidth, gCamera.m_FilmHeight);

	HandleCudaError(hipFree(pDevFrameBuffer));
}