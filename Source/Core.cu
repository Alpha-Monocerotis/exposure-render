/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"

DEVICE int* gpTracer = NULL;

typedef ExposureRender::List<ExposureRender::Volume, MAX_NO_VOLUMES>					Volumes;
typedef ExposureRender::List<ExposureRender::Light, MAX_NO_LIGHTS>						Lights;
typedef ExposureRender::List<ExposureRender::Object, MAX_NO_OBJECTS>					Objects;
typedef ExposureRender::List<ExposureRender::ClippingObject, MAX_NO_CLIPPING_OBJECTS>	ClippingObjects;
typedef ExposureRender::List<ExposureRender::Texture, MAX_NO_TEXTURES>					Textures;

DEVICE Volumes*			gpVolumes			= NULL;
DEVICE Lights*			gpLights			= NULL;
DEVICE Objects*			gpObjects			= NULL;
DEVICE ClippingObjects*	gpClippingObjects	= NULL;
DEVICE Textures*		gpTextures			= NULL;


#include "Tracer.cuh"


#include "Utilities.cuh"
/*

#include "GaussianFilter.cuh"
#include "BilateralFilter.cuh"
#include "MedianFilter.cuh"
#include "Estimate.cuh"

#include "SingleScattering.cuh"
#include "ToneMap.cuh"
#include "GradientMagnitude.cuh"
#include "AutoFocus.cuh"
*/

namespace ExposureRender
{

CudaList<Volume, MAX_NO_VOLUMES>					gVolumes("gpVolumes");
CudaList<Light, MAX_NO_LIGHTS>						gLights("gpLights");
CudaList<Object, MAX_NO_OBJECTS>					gObjects("gpObjects");
CudaList<ClippingObject, MAX_NO_CLIPPING_OBJECTS>	gClippingObjects("gpClippingObjects");
CudaList<Texture, MAX_NO_TEXTURES>					gTextures("gpTextures");

std::map<int, Tracer> gTracers;

EXPOSURE_RENDER_DLL void Resize(int TracerID, int Size[2])
{
	gTracers[TracerID].FrameBuffer.Resize(Resolution2i(Size));
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void Reset(int TracerID)
{
	gTracers[TracerID].FrameBuffer.Reset();
	gTracers[TracerID].NoIterations = 0;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void InitializeTracer(int& ID)
{
//	hipSetDevice(0);

//	gTracers.Bind(Tracer(), ID);
}

EXPOSURE_RENDER_DLL void DeinitializeTracer(int ID)
{
//	gTracers.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindVolume(Volume V, int& ID)
{
//	gSharedVolumes.Bind(Volume(V), ID);
}

EXPOSURE_RENDER_DLL void UnbindVolume(int ID)
{
//	gSharedVolumes.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindLight(Light L, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindLight(int ID)
{
//	gSharedLights.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindObject(Object O, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindObject(int ID)
{
//	gSharedObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindClippingObject(ClippingObject C, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindClippingObject(int ID)
{
//	gSharedClippingObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindTexture(Texture Texture, int& ID)
{
}

EXPOSURE_RENDER_DLL void UnbindTexture(int ID)
{
//	gSharedTextures.Unbind(ID);
}

EXPOSURE_RENDER_DLL void SetTracerVolumeIDs(int ID[MAX_NO_VOLUMES], int Size)
{
}

EXPOSURE_RENDER_DLL void SetTracerLightIDs(int ID[MAX_NO_LIGHTS], int Size)
{
}

EXPOSURE_RENDER_DLL void SetTracerObjectIDs(int ID[MAX_NO_OBJECTS], int Size)
{
}

EXPOSURE_RENDER_DLL void SetTracerClippingObjectIDs(int ID[MAX_NO_CLIPPING_OBJECTS], int Size)
{
}

EXPOSURE_RENDER_DLL void BindOpacity1D(int TracerID, ScalarTransferFunction1D Opacity1D)
{
	gTracers[TracerID].Opacity1D = Opacity1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindDiffuse1D(int TracerID, ColorTransferFunction1D Diffuse1D)
{
	gTracers[TracerID].Diffuse1D = Diffuse1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindSpecular1D(int TracerID, ColorTransferFunction1D Specular1D)
{
	gTracers[TracerID].Specular1D = Specular1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindGlossiness1D(int TracerID, ScalarTransferFunction1D Glossiness1D)
{
	gTracers[TracerID].Glossiness1D = Glossiness1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindEmission1D(int TracerID, ColorTransferFunction1D Emission1D)
{
	gTracers[TracerID].Emission1D = Emission1D;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindCamera(int TracerID, Camera Camera)
{
	gTracers[TracerID].Camera = Camera;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void BindRenderSettings(int TracerID, RenderSettings RenderSettings)
{
	gTracers[TracerID].RenderSettings = RenderSettings;
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void RenderEstimate(int TracerID)
{
//	CUDA::HostToConstantDevice(&TracerID, gActiveTracerID);

	CUDA::ThreadSynchronize();

//	SingleScattering(gTracers[TracerID].FrameBuffer.Resolution[0], gTracers[TracerID].FrameBuffer.Resolution[1]);
//	ComputeEstimate(gTracers[TracerID].FrameBuffer.Resolution[0], gTracers[TracerID].FrameBuffer.Resolution[1]);
//	ToneMap(gTracers[TracerID].FrameBuffer.Resolution[0], gTracers[TracerID].FrameBuffer.Resolution[1]);

	CUDA::ThreadSynchronize();

	gTracers[TracerID].NoIterations++; 
//	gTracers.Synchronize();
}

EXPOSURE_RENDER_DLL void GetEstimate(int TracerID, unsigned char* pData)
{
	CUDA::MemCopyDeviceToHost(gTracers[TracerID].FrameBuffer.CudaDisplayEstimate.GetPtr(), (ColorRGBAuc*)pData, gTracers[TracerID].FrameBuffer.CudaDisplayEstimate.GetNoElements());
}

EXPOSURE_RENDER_DLL void GetAutoFocusDistance(int TracerID, int FilmU, int FilmV, float& AutoFocusDistance)
{
	return;
//	ComputeAutoFocusDistance(FilmU, FilmV, AutoFocusDistance);
}

EXPOSURE_RENDER_DLL void GetNoIterations(int TracerID, int& NoIterations)
{
	NoIterations = gTracers[TracerID].NoIterations; 
}

}
