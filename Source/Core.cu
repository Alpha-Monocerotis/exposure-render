#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/reduce.h>

#include "General.cuh"

ExposureRender::KernelTimings gKernelTimings;

#include "Core.cuh"
#include "CudaUtilities.cuh"
#include "Framebuffer.cuh"
#include "Benchmark.cuh"
#include "Filter.cuh"

texture<unsigned short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexIntensity;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexGlossiness;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexEmission;

hipArray* gpIntensity	= NULL;
hipArray* gpOpacity	= NULL;
hipArray* gpDiffuse	= NULL;
hipArray* gpSpecular	= NULL;
hipArray* gpGlossiness	= NULL;
hipArray* gpEmission	= NULL;

CD ExposureRender::VolumeProperties		gVolumeProperties;
CD ExposureRender::Camera				gCamera;
CD ExposureRender::Lights				gLights;
CD ExposureRender::Clippers				gClippers;
CD ExposureRender::Reflectors			gReflectors;
CD ExposureRender::RenderSettings		gRenderSettings;
CD ExposureRender::Textures				gTextures;
CD ExposureRender::Range				gOpacityRange;
CD ExposureRender::Range				gDiffuseRange;
CD ExposureRender::Range				gSpecularRange;
CD ExposureRender::Range				gGlossinessRange;
CD ExposureRender::Range				gEmissionRange;
CD ExposureRender::GaussianFilter		gFrameEstimateFilter;
CD ExposureRender::BilateralFilter		gPostProcessingFilter;

ExposureRender::FrameBuffer				gFrameBuffer;
CD ExposureRender::Textures				gTexturesHost;

int	gNoIterations = 0;

#include "GaussianFilter.cuh"
#include "BilateralFilter.cuh"
#include "MedianFilter.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "Metropolis.cuh"
#include "ToneMap.cuh"
#include "Blend.cuh"
#include "GradientMagnitude.cuh"
#include "AutoFocus.cuh"

namespace ExposureRender
{

EXPOSURE_RENDER_DLL void Resize(int Size[2])
{
	gFrameBuffer.Resize(Resolution2i(Size));
}

EXPOSURE_RENDER_DLL void Reset()
{
	gFrameBuffer.Reset();
	gNoIterations = 0;
}

EXPOSURE_RENDER_DLL void UnbindDensityBuffer(void)
{
	CUDA::FreeArray(gpIntensity);
	CUDA::UnbindTexture(gTexIntensity);
}

EXPOSURE_RENDER_DLL void BindIntensityBuffer(unsigned short* pBuffer, int Extent[3])
{
	UnbindDensityBuffer();
	CUDA::BindTexture3D(gTexIntensity, Extent, pBuffer, gpIntensity);
}

EXPOSURE_RENDER_DLL void UnbindOpacity1D(void)
{
	CUDA::FreeArray(gpOpacity);
	CUDA::UnbindTexture(gTexOpacity);
}

EXPOSURE_RENDER_DLL void UnbindDiffuse1D(void)
{
	CUDA::FreeArray(gpDiffuse);
	CUDA::UnbindTexture(gTexDiffuse);
}

EXPOSURE_RENDER_DLL void UnbindSpecular1D(void)
{
	CUDA::FreeArray(gpSpecular);
	CUDA::UnbindTexture(gTexSpecular);
}

EXPOSURE_RENDER_DLL void UnbindGlossiness1D(void)
{
	CUDA::FreeArray(gpGlossiness);
	CUDA::UnbindTexture(gTexGlossiness);
}

EXPOSURE_RENDER_DLL void UnbindEmission1D(void)
{
	CUDA::FreeArray(gpEmission);
	CUDA::UnbindTexture(gTexEmission);
}

EXPOSURE_RENDER_DLL void BindOpacity1D(float Opacity[NO_TF_STEPS], float IntensityRange[2])
{
	UnbindOpacity1D();

	Range Int;
	Int.Set(IntensityRange);

	CUDA::HostToConstantDevice(&Int, "gOpacityRange"); 
	CUDA::BindTexture1D(gTexOpacity, NO_TF_STEPS, Opacity, gpOpacity);
}

EXPOSURE_RENDER_DLL void BindDiffuse1D(float Diffuse[3][NO_TF_STEPS], float IntensityRange[2])
{
	UnbindDiffuse1D();

	Range Int;
	Int.Set(IntensityRange);

	CUDA::HostToConstantDevice(&Int, "gDiffuseRange");

	ColorXYZAf DiffuseXYZA[NO_TF_STEPS];

	for (int i = 0; i < NO_TF_STEPS; i++)
		DiffuseXYZA[i].FromRGB(Diffuse[0][i], Diffuse[1][i], Diffuse[2][i]);

	CUDA::BindTexture1D(gTexDiffuse, NO_TF_STEPS, (float4*)DiffuseXYZA, gpDiffuse);
}

EXPOSURE_RENDER_DLL void BindSpecular1D(float Specular[3][NO_TF_STEPS], float IntensityRange[2])
{
	UnbindSpecular1D();

	Range Int;
	Int.Set(IntensityRange);

	CUDA::HostToConstantDevice(&Int, "gSpecularRange");

	ColorXYZAf SpecularXYZA[NO_TF_STEPS];

	for (int i = 0; i < NO_TF_STEPS; i++)
		SpecularXYZA[i].FromRGB(Specular[0][i], Specular[1][i], Specular[2][i]);

	CUDA::BindTexture1D(gTexSpecular, NO_TF_STEPS, (float4*)SpecularXYZA, gpSpecular);
}

EXPOSURE_RENDER_DLL void BindGlossiness1D(float Glossiness[NO_TF_STEPS], float IntensityRange[2])
{
	UnbindGlossiness1D();

	Range Int;
	Int.Set(IntensityRange);

	CUDA::HostToConstantDevice(&Int, "gGlossinessRange");
	CUDA::BindTexture1D(gTexGlossiness, NO_TF_STEPS, Glossiness, gpGlossiness);
}

EXPOSURE_RENDER_DLL void BindEmission1D(float Emission[3][NO_TF_STEPS], float IntensityRange[2])
{
	UnbindEmission1D();

	Range Int;
	Int.Set(IntensityRange);

	CUDA::HostToConstantDevice(&Int, "gEmissionRange");

	ColorXYZAf EmissionXYZA[NO_TF_STEPS];

	for (int i = 0; i < NO_TF_STEPS; i++)
		EmissionXYZA[i].FromRGB(Emission[0][i], Emission[1][i], Emission[2][i]);

	CUDA::BindTexture1D(gTexEmission, NO_TF_STEPS, (float4*)EmissionXYZA, gpEmission);
}

EXPOSURE_RENDER_DLL void BindVolumeProperties(VolumeProperties* pVolumeProperties)
{
	CUDA::HostToConstantDevice(pVolumeProperties, "gVolumeProperties");
}

EXPOSURE_RENDER_DLL void BindCamera(Camera* pCamera)
{
	const Vec3f N = Normalize(ToVec3f(pCamera->Target) - ToVec3f(pCamera->Pos));
	const Vec3f U = Normalize(Cross(N, ToVec3f(pCamera->Up)));
	const Vec3f V = Normalize(Cross(N, U));

	pCamera->N[0] = N[0];
	pCamera->N[1] = N[1];
	pCamera->N[2] = N[2];
	pCamera->U[0] = U[0];
	pCamera->U[1] = U[1];
	pCamera->U[2] = U[2];
	pCamera->V[0] = V[0];
	pCamera->V[1] = V[1];
	pCamera->V[2] = V[2];

	if (pCamera->FocalDistance == -1.0f)
		pCamera->FocalDistance = (ToVec3f(pCamera->Target) - ToVec3f(pCamera->Pos)).Length();

	float Scale = 0.0f;

	Scale = tanf((0.5f * pCamera->FOV / RAD_F));

	const float AspectRatio = (float)pCamera->FilmHeight / (float)pCamera->FilmWidth;

	if (AspectRatio > 1.0f)
	{
		pCamera->Screen[0][0] = -Scale;
		pCamera->Screen[0][1] = Scale;
		pCamera->Screen[1][0] = -Scale * AspectRatio;
		pCamera->Screen[1][1] = Scale * AspectRatio;
	}
	else
	{
		pCamera->Screen[0][0] = -Scale / AspectRatio;
		pCamera->Screen[0][1] = Scale / AspectRatio;
		pCamera->Screen[1][0] = -Scale;
		pCamera->Screen[1][1] = Scale;
	}

	pCamera->InvScreen[0] = (pCamera->Screen[0][1] - pCamera->Screen[0][0]) / (float)pCamera->FilmWidth;
	pCamera->InvScreen[1] = (pCamera->Screen[1][1] - pCamera->Screen[1][0]) / (float)pCamera->FilmHeight;

	CUDA::HostToConstantDevice(pCamera, "gCamera");
}

EXPOSURE_RENDER_DLL void BindLights(Lights* pLights)
{
	CUDA::HostToConstantDevice(pLights, "gLights");
}

EXPOSURE_RENDER_DLL void BindClippers(Clippers* pClippers)
{
	CUDA::HostToConstantDevice(pClippers, "gClippers");
}

EXPOSURE_RENDER_DLL void BindReflectors(Reflectors* pReflectors)
{
	CUDA::HostToConstantDevice(pReflectors, "gReflectors");
}

EXPOSURE_RENDER_DLL void BindRenderSettings(RenderSettings* pRenderSettings)
{
	CUDA::HostToConstantDevice(pRenderSettings, "gRenderSettings");
}

EXPOSURE_RENDER_DLL void BindFiltering(Filtering* pFiltering)
{
	// Frame estimate filter
	GaussianFilter Gaussian;
	
	Gaussian.KernelRadius = pFiltering->FrameEstimateFilter.KernelRadius;

	const int KernelSize = (2 * Gaussian.KernelRadius) + 1;

	for (int i = 0; i < KernelSize; i++)
		Gaussian.KernelD[i] = Gauss2D(pFiltering->FrameEstimateFilter.Sigma, Gaussian.KernelRadius - i, 0);

	CUDA::HostToConstantDevice(&Gaussian, "gFrameEstimateFilter");

	// Post processing filter
	BilateralFilter Bilateral;

	const int SigmaMax = (int)max(pFiltering->PostProcessingFilter.SigmaD, pFiltering->PostProcessingFilter.SigmaR);
	
	Bilateral.KernelRadius = (int)ceilf(2.0f * (float)SigmaMax);  

	const float TwoSigmaRSquared = 2 * pFiltering->PostProcessingFilter.SigmaR * pFiltering->PostProcessingFilter.SigmaR;

	const int kernelSize = Bilateral.KernelRadius * 2 + 1;
	const int center = (kernelSize - 1) / 2;

	for (int x = -center; x < -center + kernelSize; x++)
		Bilateral.KernelD[x + center] = Gauss2D(pFiltering->PostProcessingFilter.SigmaD, x, 0);

	for (int i = 0; i < 256; i++)
		Bilateral.GaussSimilarity[i] = expf(-((float)i / TwoSigmaRSquared));

	CUDA::HostToConstantDevice(&Bilateral, "gPostProcessingFilter");
}

EXPOSURE_RENDER_DLL void BindTextures(Textures* pTextures)
{
	if (gTexturesHost.NoTextures + 1 >= MAX_NO_TEXTURES)
		throw(Exception("Texture Error", "Maximum no. textures reached"));
	/*
	Texture& T = gTexturesHost.TextureList[gTexturesHost.NoTextures];

	T = *pTexture;

	if (Type == 0)
	{
		CUDA::MemCopyHostToDevice(pTexture->Image.pData, T.Image.pData, pTexture.Image.Size[0] * pTexture.Image.Size[1] * 3);
	}
	*/


	// Iterate over all textures and see 
}

EXPOSURE_RENDER_DLL void RenderEstimate()
{
	gKernelTimings.Reset();

	FrameBuffer* pDevFrameBuffer = NULL;

	CUDA::Allocate(pDevFrameBuffer);
	CUDA::MemCopyHostToDevice(&gFrameBuffer, pDevFrameBuffer);

	CUDA::ThreadSynchronize();

	SingleScattering(pDevFrameBuffer, gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1]);
	FilterGaussian(gFrameBuffer.CudaFrameEstimate.GetPtr(), gFrameBuffer.CudaFrameEstimateTemp.GetPtr(), gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1]);
	ComputeEstimate(pDevFrameBuffer, gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1]);
	ToneMap(pDevFrameBuffer, gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1]);
//	FilterBilateral(gFrameBuffer.CudaDisplayEstimate.GetPtr(), gFrameBuffer.CudaDisplayEstimateTemp.GetPtr(), gFrameBuffer.CudaDisplayEstimateFiltered.GetPtr(), gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1]);
//	MedianFilter(gFrameBuffer.CudaDisplayEstimate.GetPtr(), gFrameBuffer.CudaDisplayEstimateFiltered.GetPtr(), gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1]);
//	Blend(gFrameBuffer.CudaDisplayEstimate.GetPtr(), gFrameBuffer.CudaDisplayEstimateFiltered.GetPtr(), gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1]);

	CUDA::Free(pDevFrameBuffer);

	CUDA::ThreadSynchronize();

	gNoIterations++;
}

EXPOSURE_RENDER_DLL void GetEstimate(unsigned char* pData)
{
	CUDA::MemCopyDeviceToHost(gFrameBuffer.CudaDisplayEstimate.GetPtr(), (ColorRGBAuc*)pData, gFrameBuffer.CudaDisplayEstimate.GetNoElements());
}

EXPOSURE_RENDER_DLL void RecordBenchmarkImage()
{
//	CUDA::MemCopyDeviceToDevice(gFrameBuffer.CudaDisplayEstimate.GetPtr(), gFrameBuffer.BenchmarkEstimateRgbaLdr.GetPtr(), gFrameBuffer.CudaDisplayEstimate.GetNoElements()); 
}

EXPOSURE_RENDER_DLL void GetAverageNrmsError(float& AverageNrmsError)
{
	FrameBuffer* pDevFrameBuffer = NULL;
	CUDA::Allocate(pDevFrameBuffer);
	CUDA::MemCopyHostToDevice(&gFrameBuffer, pDevFrameBuffer);

	ComputeAverageNrmsError(gFrameBuffer, pDevFrameBuffer, gFrameBuffer.Resolution[0], gFrameBuffer.Resolution[1], AverageNrmsError);

	CUDA::Free(pDevFrameBuffer);
}

EXPOSURE_RENDER_DLL void GetMaximumGradientMagnitude(float& MaximumGradientMagnitude, int Extent[3])
{
	ComputeGradientMagnitudeVolume(Extent, MaximumGradientMagnitude);
}

EXPOSURE_RENDER_DLL void GetAutoFocusDistance(int FilmU, int FilmV, float& AutoFocusDistance)
{
	ComputeAutoFocusDistance(FilmU, FilmV, AutoFocusDistance);
}

EXPOSURE_RENDER_DLL void GetKernelTimings(KernelTimings* pKernelTimings)
{
	if (!pKernelTimings)
		return;

	*pKernelTimings = gKernelTimings;
}

EXPOSURE_RENDER_DLL void GetMemoryUsed(float& MemoryUsed)
{
	/*
	CUsize_t free = 0;
    CUsize_t total = 0;
    hipMemGetInfo(&free, &total);
    return total - free;
	*/ 
}

EXPOSURE_RENDER_DLL void GetNoIterations(int& NoIterations)
{
	NoIterations = gNoIterations; 
}

EXPOSURE_RENDER_DLL void Deinitialize()
{
	UnbindDensityBuffer();
	UnbindOpacity1D();
	UnbindDiffuse1D();
	UnbindSpecular1D();
	UnbindGlossiness1D();
	UnbindEmission1D();

	gFrameBuffer.Free();
}

EXPOSURE_RENDER_DLL void Initialize()
{
	Deinitialize();
}

}