#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"

#include "Slicing.cuh"
#include "VolumeInfo.cuh"
#include "RenderInfo.cuh"
#include "Lighting.cuh"

texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, hipTextureType3D, hipReadModeElementType>			gTexExtinction;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexGlossiness;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexIOR;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipChannelFormatDesc gFloatChannelDesc = hipCreateChannelDesc<float>();
hipChannelFormatDesc gFloat4ChannelDesc = hipCreateChannelDesc<float4>();

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
hipArray* gpOpacityArray				= NULL;
hipArray* gpDiffuseArray				= NULL;
hipArray* gpSpecularArray				= NULL;
hipArray* gpGlossinessArray			= NULL;
hipArray* gpIORArray					= NULL;
hipArray* gpEmissionArray				= NULL;

CD VolumeInfo	gVolumeInfo;
CD Lighting		gLighting;
CD Slicing		gSlicing;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

void BindIntensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();

	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexGradientMagnitude));
}

void BindTransferFunctions1D(float Opacity[128], float Diffuse[3][128], float Specular[3][128], float Glossiness[128], float IOR[128], float Emission[3][128], int N)
{
	// Opacity
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, N * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, gFloatChannelDesc));

	// Diffuse
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pDiffuseXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pDiffuseXYZA[i].FromRGB(Diffuse[0][i], Diffuse[1][i], Diffuse[2][i]);

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, pDiffuseXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, gFloat4ChannelDesc));

	delete[] pDiffuseXYZA;

	// Specular
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pSpecularXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pSpecularXYZA[i].FromRGB(Specular[0][i], Specular[1][i], Specular[2][i]);

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, pSpecularXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, gFloat4ChannelDesc));

	delete[] pSpecularXYZA;

	// Glossiness
	gTexGlossiness.normalized		= true;
	gTexGlossiness.filterMode		= hipFilterModeLinear;
	gTexGlossiness.addressMode[0]	= hipAddressModeClamp;

	if (gpGlossinessArray == NULL)
		HandleCudaError(hipMallocArray(&gpGlossinessArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpGlossinessArray, 0, 0, Glossiness, N * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexGlossiness, gpGlossinessArray, gFloatChannelDesc));

	// IOR
	gTexIOR.normalized		= true;
	gTexIOR.filterMode		= hipFilterModeLinear;
	gTexIOR.addressMode[0]	= hipAddressModeClamp;

	if (gpIORArray == NULL)
		HandleCudaError(hipMallocArray(&gpIORArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpIORArray, 0, 0, IOR, N * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexIOR, gpIORArray, gFloatChannelDesc));

	// Emission
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pEmissionXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pEmissionXYZA[i].FromRGB(Emission[0][i], Emission[1][i], Emission[2][i]);

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, pEmissionXYZA, N * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, gFloat4ChannelDesc));

	delete[] pEmissionXYZA;
}

void UnbindTransferFunctions1D(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	HandleCudaError(hipFreeArray(gpSpecularArray));
	HandleCudaError(hipFreeArray(gpGlossinessArray));
	HandleCudaError(hipFreeArray(gpEmissionArray));

	gpOpacityArray		= NULL;
	gpDiffuseArray		= NULL;
	gpSpecularArray		= NULL;
	gpGlossinessArray	= NULL;
	gpEmissionArray		= NULL;

	HandleCudaError(hipUnbindTexture(gTexOpacity));
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
	HandleCudaError(hipUnbindTexture(gTexSpecular));
	HandleCudaError(hipUnbindTexture(gTexGlossiness));
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void RenderEstimate(VolumeInfo* pVolumeInfo, RenderInfo* pRenderInfo, Lighting* pLighting, Slicing* pSlicing, FrameBuffer* pFrameBuffer)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gVolumeInfo"), pVolumeInfo, sizeof(VolumeInfo)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gLighting"), pLighting, sizeof(Lighting)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gSlicing"), pSlicing, sizeof(Slicing)));
	
	RenderInfo*		pDevRenderInfo	= NULL;
	FrameBuffer*	pDevFrameBuffer	= NULL;

	HandleCudaError(hipMalloc(&pDevRenderInfo, sizeof(RenderInfo)));
	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));

	HandleCudaError(hipMemcpy(pDevRenderInfo, pRenderInfo, sizeof(RenderInfo), hipMemcpyHostToDevice));
	HandleCudaError(hipMemcpy(pDevFrameBuffer, pFrameBuffer, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	const dim3 BlockDim(8, 8);
	const dim3 GridDim((int)ceilf((float)pRenderInfo->m_FilmWidth / (float)BlockDim.x), (int)ceilf((float)pRenderInfo->m_FilmHeight / (float)BlockDim.y));

	SingleScattering(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
	BlurEstimate(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
	ComputeEstimate(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
	ToneMap(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
//	Denoise(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);

	HandleCudaError(hipFree(pDevRenderInfo));
	HandleCudaError(hipFree(pDevFrameBuffer));
}