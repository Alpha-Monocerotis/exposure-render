#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"

#include "Slice.cuh"

#include "VolumeInfo.cuh"
#include "RenderInfo.cuh"

texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexDensity;
texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<float, hipTextureType3D, hipReadModeElementType>			gTexExtinction;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexRoughness;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipChannelFormatDesc gFloatChannelDesc = hipCreateChannelDesc<float>();
hipChannelFormatDesc gFloat4ChannelDesc = hipCreateChannelDesc<float4>();

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
hipArray* gpOpacityArray				= NULL;
hipArray* gpDiffuseArray				= NULL;
hipArray* gpSpecularArray				= NULL;
hipArray* gpRoughnessArray				= NULL;
hipArray* gpEmissionArray				= NULL;

CD VolumeInfo	gVolumeInfo;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

void BindIntensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();

	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexDensity.normalized		= true;
	gTexDensity.filterMode		= hipFilterModeLinear;      
	gTexDensity.addressMode[0]	= hipAddressModeClamp;  
	gTexDensity.addressMode[1]	= hipAddressModeClamp;
  	gTexDensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexDensity, gpDensityArray, ChannelDesc));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexGradientMagnitude));
}

void BindTransferFunctions1D(float* pOpacity, float* pDiffuse, float* pSpecular, float* pRoughness, float* pEmission, int N)
{
	// Opacity
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, pOpacity, N * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, gFloatChannelDesc));

	// Diffuse
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pDiffuseXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pDiffuseXYZA[i].FromRGB(pDiffuse[i * 3], pDiffuse[i * 3 + 1], pDiffuse[i * 3 + 2]);

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, pDiffuseXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, gFloat4ChannelDesc));

	delete[] pDiffuseXYZA;

	// Specular
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pSpecularXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pSpecularXYZA[i].FromRGB(pSpecular[i * 3], pSpecular[i * 3 + 1], pSpecular[i * 3 + 2]);

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, pSpecularXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, gFloat4ChannelDesc));

	delete[] pSpecularXYZA;

	// Roughness
	gTexRoughness.normalized		= true;
	gTexRoughness.filterMode		= hipFilterModeLinear;
	gTexRoughness.addressMode[0]	= hipAddressModeClamp;

	if (gpRoughnessArray == NULL)
		HandleCudaError(hipMallocArray(&gpRoughnessArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpRoughnessArray, 0, 0, pRoughness, N * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexRoughness, gpRoughnessArray, gFloatChannelDesc));

	// Emission
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pEmissionXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pEmissionXYZA[i].FromRGB(pEmission[i * 3], pEmission[i * 3 + 1], pEmission[i * 3 + 2]);

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, pEmissionXYZA, N * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, gFloat4ChannelDesc));

	delete[] pEmissionXYZA;
}

void UnbindTransferFunctions1D(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	HandleCudaError(hipFreeArray(gpSpecularArray));
	HandleCudaError(hipFreeArray(gpRoughnessArray));
	HandleCudaError(hipFreeArray(gpEmissionArray));

	gpOpacityArray		= NULL;
	gpDiffuseArray		= NULL;
	gpSpecularArray		= NULL;
	gpRoughnessArray	= NULL;
	gpEmissionArray		= NULL;

	HandleCudaError(hipUnbindTexture(gTexOpacity));
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
	HandleCudaError(hipUnbindTexture(gTexSpecular));
	HandleCudaError(hipUnbindTexture(gTexRoughness));
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void RenderEstimate(VolumeInfo* pVolumeInfo, RenderInfo* pRenderInfo, FrameBuffer* pFrameBuffer)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gVolumeInfo"), pVolumeInfo, sizeof(VolumeInfo)));

	RenderInfo*		pDevRenderInfo	= NULL;
	FrameBuffer*	pDevFrameBuffer	= NULL;

	HandleCudaError(hipMalloc(&pDevRenderInfo, sizeof(RenderInfo)));
	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));

	HandleCudaError(hipMemcpy(pDevRenderInfo, pRenderInfo, sizeof(RenderInfo), hipMemcpyHostToDevice));
	HandleCudaError(hipMemcpy(pDevFrameBuffer, pFrameBuffer, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	const dim3 BlockDim(8, 8);
	const dim3 GridDim((int)ceilf((float)pRenderInfo->m_FilmWidth / (float)BlockDim.x), (int)ceilf((float)pRenderInfo->m_FilmHeight / (float)BlockDim.y));

	SingleScattering(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
//	BlurEstimate(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
	ComputeEstimate(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
	ToneMap(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);
//	Denoise(pDevRenderInfo, pDevFrameBuffer, pRenderInfo->m_FilmWidth, pRenderInfo->m_FilmHeight);

	HandleCudaError(hipFree(pDevRenderInfo));
	HandleCudaError(hipFree(pDevFrameBuffer));
}