#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"
#include "General.cuh"
#include "Framebuffer.cuh"
#include "Benchmark.cuh"

texture<unsigned short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexIntensity;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>					gTexGlossiness;
texture<float4, hipTextureType1D, hipReadModeElementType>					gTexEmission;

hipChannelFormatDesc gFloatChannelDesc = hipCreateChannelDesc<float>();
hipChannelFormatDesc gFloat4ChannelDesc = hipCreateChannelDesc<float4>();

hipArray* gpIntensity			= NULL;
hipArray* gpOpacity			= NULL;
hipArray* gpDiffuse			= NULL;
hipArray* gpSpecular			= NULL;
hipArray* gpGlossiness			= NULL;
hipArray* gpEmission			= NULL;

CD ErVolume			gVolume;
CD ErCamera			gCamera;
CD ErLights			gLights;
CD ErClippers		gClippers;
CD ErReflectors		gReflectors;
CD ErDenoise		gDenoise;
CD ErScattering		gScattering;
CD ErBlur			gBlur;
CD ErRange			gOpacityRange;
CD ErRange			gDiffuseRange;
CD ErRange			gSpecularRange;
CD ErRange			gGlossinessRange;
CD ErRange			gEmissionRange;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "ToneMap.cuh"
#include "GradientMagnitude.cuh"

FrameBuffer FB;

void ErInitialize()
{
}

void ErDeinitialize()
{
	ErUnbindDensityBuffer();
	ErUnbindOpacity1D();
	ErUnbindDiffuse1D();
	ErUnbindSpecular1D();
	ErUnbindGlossiness1D();
	ErUnbindEmission1D();
}

void ErResize(int Size[2])
{
	FB.Resize(Resolution2i(Size));
}

void ErResetFrameBuffer()
{
	FB.Reset();
}

void ErBindIntensityBuffer(unsigned short* pBuffer, int Extent[3])
{
	ErUnbindDensityBuffer();

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<unsigned short>();

	hipExtent CudaExtent = make_hipExtent(Extent[0], Extent[1], Extent[2]);

	hipMalloc3DArray(&gpIntensity, &ChannelDesc, CudaExtent);

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr		= make_hipPitchedPtr(pBuffer, CudaExtent.width * sizeof(unsigned short), CudaExtent.width, CudaExtent.height);
	CopyParams.dstArray		= gpIntensity;
	CopyParams.extent		= CudaExtent;
	CopyParams.kind			= hipMemcpyHostToDevice;
	
	hipMemcpy3D(&CopyParams);

	gTexIntensity.normalized		= true;
	gTexIntensity.filterMode		= hipFilterModeLinear;      
	gTexIntensity.addressMode[0]	= hipAddressModeClamp;  
	gTexIntensity.addressMode[1]	= hipAddressModeClamp;
  	gTexIntensity.addressMode[2]	= hipAddressModeClamp;

	hipBindTextureToArray(gTexIntensity, gpIntensity, ChannelDesc);
}

void ErUnbindDensityBuffer(void)
{
	hipFreeArray(gpIntensity);
	gpIntensity = NULL;
	hipUnbindTexture(gTexIntensity);
}

void ErBindOpacity1D(float Opacity[NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	hipMemcpyToSymbol(HIP_SYMBOL("gOpacityRange"), &Int, sizeof(ErRange));

	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	if (gpOpacity == NULL)
		hipMallocArray(&gpOpacity, &gFloatChannelDesc, NO_GRADIENT_STEPS, 1);

	hipMemcpyToArray(gpOpacity, 0, 0, Opacity, NO_GRADIENT_STEPS * sizeof(float), hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexOpacity, gpOpacity, gFloatChannelDesc);
}

void ErBindDiffuse1D(float Diffuse[3][NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	hipMemcpyToSymbol(HIP_SYMBOL("gDiffuseRange"), &Int, sizeof(ErRange));

	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	if (gpDiffuse == NULL)
		hipMallocArray(&gpDiffuse, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1);

	ColorXYZAf* pDiffuseXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pDiffuseXYZA[i].FromRGB(Diffuse[0][i], Diffuse[1][i], Diffuse[2][i]);

	hipMemcpyToArray(gpDiffuse, 0, 0, pDiffuseXYZA, NO_GRADIENT_STEPS * sizeof(float4), hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexDiffuse, gpDiffuse, gFloat4ChannelDesc);

	delete[] pDiffuseXYZA;
}

void ErBindSpecular1D(float Specular[3][NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	hipMemcpyToSymbol(HIP_SYMBOL("gSpecularRange"), &Int, sizeof(ErRange));

	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	if (gpSpecular == NULL)
		hipMallocArray(&gpSpecular, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1);

	ColorXYZAf* pSpecularXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pSpecularXYZA[i].FromRGB(Specular[0][i], Specular[1][i], Specular[2][i]);

	hipMemcpyToArray(gpSpecular, 0, 0, pSpecularXYZA, NO_GRADIENT_STEPS * sizeof(float4), hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexSpecular, gpSpecular, gFloat4ChannelDesc);

	delete[] pSpecularXYZA;
}

void ErBindGlossiness1D(float Glossiness[NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	hipMemcpyToSymbol(HIP_SYMBOL("gGlossinessRange"), &Int, sizeof(ErRange));

	gTexGlossiness.normalized		= true;
	gTexGlossiness.filterMode		= hipFilterModeLinear;
	gTexGlossiness.addressMode[0]	= hipAddressModeClamp;

	if (gpGlossiness == NULL)
		hipMallocArray(&gpGlossiness, &gFloatChannelDesc, NO_GRADIENT_STEPS, 1);

	hipMemcpyToArray(gpGlossiness, 0, 0, Glossiness, NO_GRADIENT_STEPS * sizeof(float),  hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexGlossiness, gpGlossiness, gFloatChannelDesc);
}

void ErBindEmission1D(float Emission[3][NO_GRADIENT_STEPS], float Range[2])
{
	ErRange Int;
	Int.Set(Range);

	hipMemcpyToSymbol(HIP_SYMBOL("gEmissionRange"), &Int, sizeof(ErRange));

	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	if (gpEmission == NULL)
		hipMallocArray(&gpEmission, &gFloat4ChannelDesc, NO_GRADIENT_STEPS, 1);

	ColorXYZAf* pEmissionXYZA = new ColorXYZAf[NO_GRADIENT_STEPS];

	for (int i = 0; i < NO_GRADIENT_STEPS; i++)
		pEmissionXYZA[i].FromRGB(Emission[0][i], Emission[1][i], Emission[2][i]);

	hipMemcpyToArray(gpEmission, 0, 0, pEmissionXYZA, NO_GRADIENT_STEPS * sizeof(float4),  hipMemcpyHostToDevice);
	hipBindTextureToArray(gTexEmission, gpEmission, gFloat4ChannelDesc);

	delete[] pEmissionXYZA;
}

void ErUnbindOpacity1D(void)
{
	hipFreeArray(gpOpacity);
	gpOpacity = NULL;
	hipUnbindTexture(gTexOpacity);
}

void ErUnbindDiffuse1D(void)
{
	hipFreeArray(gpDiffuse);
	gpDiffuse = NULL;
	hipUnbindTexture(gTexDiffuse);
}

void ErUnbindSpecular1D(void)
{
	hipFreeArray(gpSpecular);
	gpSpecular	= NULL;
	hipUnbindTexture(gTexSpecular);
}

void ErUnbindGlossiness1D(void)
{
	hipFreeArray(gpGlossiness);
	gpGlossiness = NULL;
	hipUnbindTexture(gTexGlossiness);
}

void ErUnbindEmission1D(void)
{
	hipFreeArray(gpEmission);
	gpEmission	= NULL;
	hipUnbindTexture(gTexEmission);
}

void ErBindVolume(ErVolume* pVolume)
{
	hipMemcpyToSymbol(HIP_SYMBOL("gVolume"), pVolume, sizeof(ErVolume));
}

void ErBindCamera(ErCamera* pCamera)
{
	const Vec3f N = Normalize(ToVec3f(pCamera->Target) - ToVec3f(pCamera->Pos));
	const Vec3f U = Normalize(Cross(N, ToVec3f(pCamera->Up)));
	const Vec3f V = Normalize(Cross(N, U));

	pCamera->N[0] = N[0];
	pCamera->N[1] = N[1];
	pCamera->N[2] = N[2];
	pCamera->U[0] = U[0];
	pCamera->U[1] = U[1];
	pCamera->U[2] = U[2];
	pCamera->V[0] = V[0];
	pCamera->V[1] = V[1];
	pCamera->V[2] = V[2];

	float Scale = 0.0f;

	Scale = tanf((0.5f * pCamera->FOV / RAD_F));

	const float AspectRatio = (float)pCamera->FilmHeight / (float)pCamera->FilmWidth;

	if (AspectRatio > 1.0f)
	{
		pCamera->Screen[0][0] = -Scale;
		pCamera->Screen[0][1] = Scale;
		pCamera->Screen[1][0] = -Scale * AspectRatio;
		pCamera->Screen[1][1] = Scale * AspectRatio;
	}
	else
	{
		pCamera->Screen[0][0] = -Scale / AspectRatio;
		pCamera->Screen[0][1] = Scale / AspectRatio;
		pCamera->Screen[1][0] = -Scale;
		pCamera->Screen[1][1] = Scale;
	}

	pCamera->InvScreen[0] = (pCamera->Screen[0][1] - pCamera->Screen[0][0]) / (float)pCamera->FilmWidth;
	pCamera->InvScreen[1] = (pCamera->Screen[1][1] - pCamera->Screen[1][0]) / (float)pCamera->FilmHeight;

	hipMemcpyToSymbol(HIP_SYMBOL("gCamera"), pCamera, sizeof(ErCamera));
}

void ErBindLights(ErLights* pLights)
{
	hipMemcpyToSymbol(HIP_SYMBOL("gLights"), pLights, sizeof(ErLights));
}

void ErBindClippers(ErClippers* pClippers)
{
	hipMemcpyToSymbol(HIP_SYMBOL("gClippers"), pClippers, sizeof(ErClippers));
}

void ErBindReflectors(ErReflectors* pReflectors)
{
	hipMemcpyToSymbol(HIP_SYMBOL("gReflectors"), pReflectors, sizeof(ErReflectors));
}

void ErBindDenoise(ErDenoise* pDenoise)
{
	hipMemcpyToSymbol(HIP_SYMBOL("gDenoise"), pDenoise, sizeof(ErDenoise));
}

void ErBindScattering(ErScattering* pScattering)
{
	hipMemcpyToSymbol(HIP_SYMBOL("gScattering"), pScattering, sizeof(ErScattering));
}

void ErBindBlur(ErBlur* pBlur)
{
	hipMemcpyToSymbol(HIP_SYMBOL("gBlur"), pBlur, sizeof(ErBlur));
}

void ErRenderEstimate()
{
	FrameBuffer* pDevFrameBuffer = NULL;
	hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer));
	hipMemcpy(pDevFrameBuffer, &FB, sizeof(FrameBuffer), hipMemcpyHostToDevice);

	SingleScattering(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);
	BlurEstimate(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);
	ComputeEstimate(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);
	ToneMap(pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1]);

	hipFree(pDevFrameBuffer);
}

void ErGetEstimate(unsigned char* pData)
{
	hipMemcpy(FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToHost);
	memcpy(pData, FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize());
}

void ErGetFrameEstimate(unsigned char* pData)
{
	//hipMemcpy(FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToHost);
	//memcpy(pData, FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize());
}

void ErGetDepthBuffer(unsigned char* pData)
{
	//hipMemcpy(FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToHost);
	//memcpy(pData, FB.HostDisplayEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize());
}

void ErRecordBenchmarkImage()
{
	hipMemcpy(FB.BenchmarkEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetPtr(), FB.CudaRunningEstimateRgbaLdr.GetSize(), hipMemcpyDeviceToDevice);
}

void ErGetAverageNrmsError(float& AverageNrmsError)
{
	FrameBuffer* pDevFrameBuffer = NULL;
	hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer));
	hipMemcpy(pDevFrameBuffer, &FB, sizeof(FrameBuffer), hipMemcpyHostToDevice);

	ComputeAverageNrmsError(FB, pDevFrameBuffer, FB.Resolution[0], FB.Resolution[1], AverageNrmsError);

	hipFree(pDevFrameBuffer);
}

void ErGetMaximumGradientMagnitude(float& MaximumGradientMagnitude, int Extent[3])
{
	ComputeGradientMagnitudeVolume(Extent, MaximumGradientMagnitude);
}