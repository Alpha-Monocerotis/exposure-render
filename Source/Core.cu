#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "Core.cuh"

#include "Slicing.cuh"
#include "VolumeInfo.cuh"
#include "RenderInfo.cuh"
#include "Lighting.cuh"

texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexIntensity;
texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexGradientMagnitude;
texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexExtinction;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexGlossiness;
texture<float, hipTextureType1D, hipReadModeElementType>			gTexIOR;
texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipChannelFormatDesc gFloatChannelDesc = hipCreateChannelDesc<float>();
hipChannelFormatDesc gFloat4ChannelDesc = hipCreateChannelDesc<float4>();

hipArray* gpDensityArray			= NULL;
hipArray* gpExtinction				= NULL;
hipArray* gpGradientMagnitudeArray	= NULL;
hipArray* gpOpacityArray			= NULL;
hipArray* gpDiffuseArray			= NULL;
hipArray* gpSpecularArray			= NULL;
hipArray* gpGlossinessArray		= NULL;
hipArray* gpIORArray				= NULL;
hipArray* gpEmissionArray			= NULL;

CD Volume		gVolume;
CD Camera		gCamera;
CD Lighting		gLighting;
CD Slicing		gSlicing;
CD Denoise		gDenoise;
CD Scattering	gScattering;
CD Blur			gBlur;

#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

void BindIntensityBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();

	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpDensityArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexIntensity.normalized		= true;
	gTexIntensity.filterMode		= hipFilterModeLinear;      
	gTexIntensity.addressMode[0]	= hipAddressModeClamp;  
	gTexIntensity.addressMode[1]	= hipAddressModeClamp;
  	gTexIntensity.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexIntensity, gpDensityArray, ChannelDesc));
}

void BindExtinction(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();

	HandleCudaError(hipMalloc3DArray(&gpExtinction, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpExtinction;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexExtinction.normalized		= true;
	gTexExtinction.filterMode		= hipFilterModePoint;      
	gTexExtinction.addressMode[0]	= hipAddressModeClamp;  
	gTexExtinction.addressMode[1]	= hipAddressModeClamp;
  	gTexExtinction.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexExtinction, gpExtinction, ChannelDesc));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent Extent)
{
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<short>();
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &ChannelDesc, Extent));

	hipMemcpy3DParms CopyParams = {0};

	CopyParams.srcPtr	= make_hipPitchedPtr(pBuffer, Extent.width * sizeof(short), Extent.width, Extent.height);
	CopyParams.dstArray	= gpGradientMagnitudeArray;
	CopyParams.extent	= Extent;
	CopyParams.kind		= hipMemcpyHostToDevice;
	
	HandleCudaError(hipMemcpy3D(&CopyParams));

	gTexGradientMagnitude.normalized		= true;
	gTexGradientMagnitude.filterMode		= hipFilterModeLinear;      
	gTexGradientMagnitude.addressMode[0]	= hipAddressModeClamp;  
	gTexGradientMagnitude.addressMode[1]	= hipAddressModeClamp;
  	gTexGradientMagnitude.addressMode[2]	= hipAddressModeClamp;

	HandleCudaError(hipBindTextureToArray(gTexGradientMagnitude, gpGradientMagnitudeArray, ChannelDesc));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexIntensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexGradientMagnitude));
}

void BindTransferFunctions1D(float Opacity[128], float Diffuse[3][128], float Specular[3][128], float Glossiness[128], float IOR[128], float Emission[3][128], int N)
{
	// Opacity
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, N * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, gFloatChannelDesc));

	// Diffuse
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pDiffuseXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pDiffuseXYZA[i].FromRGB(Diffuse[0][i], Diffuse[1][i], Diffuse[2][i]);

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, pDiffuseXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, gFloat4ChannelDesc));

	delete[] pDiffuseXYZA;

	// Specular
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pSpecularXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pSpecularXYZA[i].FromRGB(Specular[0][i], Specular[1][i], Specular[2][i]);

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, pSpecularXYZA, N * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, gFloat4ChannelDesc));

	delete[] pSpecularXYZA;

	// Glossiness
	gTexGlossiness.normalized		= true;
	gTexGlossiness.filterMode		= hipFilterModeLinear;
	gTexGlossiness.addressMode[0]	= hipAddressModeClamp;

	if (gpGlossinessArray == NULL)
		HandleCudaError(hipMallocArray(&gpGlossinessArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpGlossinessArray, 0, 0, Glossiness, N * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexGlossiness, gpGlossinessArray, gFloatChannelDesc));

	// IOR
	gTexIOR.normalized		= true;
	gTexIOR.filterMode		= hipFilterModeLinear;
	gTexIOR.addressMode[0]	= hipAddressModeClamp;

	if (gpIORArray == NULL)
		HandleCudaError(hipMallocArray(&gpIORArray, &gFloatChannelDesc, N, 1));

	HandleCudaError(hipMemcpyToArray(gpIORArray, 0, 0, IOR, N * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexIOR, gpIORArray, gFloatChannelDesc));

	// Emission
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &gFloat4ChannelDesc, N, 1));

	ColorXYZAf* pEmissionXYZA = new ColorXYZAf[N];

	for (int i = 0; i < N; i++)
		pEmissionXYZA[i].FromRGB(Emission[0][i], Emission[1][i], Emission[2][i]);

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, pEmissionXYZA, N * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, gFloat4ChannelDesc));

	delete[] pEmissionXYZA;
}

void UnbindTransferFunctions1D(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	HandleCudaError(hipFreeArray(gpSpecularArray));
	HandleCudaError(hipFreeArray(gpGlossinessArray));
	HandleCudaError(hipFreeArray(gpEmissionArray));

	gpOpacityArray		= NULL;
	gpDiffuseArray		= NULL;
	gpSpecularArray		= NULL;
	gpGlossinessArray	= NULL;
	gpEmissionArray		= NULL;

	HandleCudaError(hipUnbindTexture(gTexOpacity));
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
	HandleCudaError(hipUnbindTexture(gTexSpecular));
	HandleCudaError(hipUnbindTexture(gTexGlossiness));
	HandleCudaError(hipUnbindTexture(gTexEmission));
}

void RenderEstimate(Volume* pVolume, Camera* pCamera, Lighting* pLighting, Slicing* pSlicing, Denoise* pDenoise, Scattering* pScattering, Blur* pBlur, FrameBuffer* pFrameBuffer)
{
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gVolume"), pVolume, sizeof(Volume)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gCamera"), pCamera, sizeof(Camera)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gLighting"), pLighting, sizeof(Lighting)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gSlicing"), pSlicing, sizeof(Slicing)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gDenoise"), pDenoise, sizeof(Denoise)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gScattering"), pScattering, sizeof(Scattering)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL("gBlur"), pBlur, sizeof(Blur)));
	
	FrameBuffer* pDevFrameBuffer = NULL;

	HandleCudaError(hipMalloc(&pDevFrameBuffer, sizeof(FrameBuffer)));

	HandleCudaError(hipMemcpy(pDevFrameBuffer, pFrameBuffer, sizeof(FrameBuffer), hipMemcpyHostToDevice));

	const dim3 BlockDim(8, 8);
	const dim3 GridDim((int)ceilf((float)gCamera.m_FilmWidth / (float)BlockDim.x), (int)ceilf((float)gCamera.m_FilmHeight / (float)BlockDim.y));

	SingleScattering(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
	BlurEstimate(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
	ComputeEstimate(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
	ToneMap(pDevFrameBuffer, pCamera->m_FilmWidth, pCamera->m_FilmHeight);
//	ReduceNoise(pDevRenderInfo, pDevFrameBuffer, gCamera.m_FilmWidth, gCamera.m_FilmHeight);

	HandleCudaError(hipFree(pDevFrameBuffer));
}