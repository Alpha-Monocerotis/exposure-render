/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#define __CUDA_ARCH__ 200

#include "tracer.h"
#include "volume.h"
#include "light.h"
#include "object.h"
#include "clippingobject.h"
#include "texture.h"
#include "bitmap.h"

DEVICE ExposureRender::Tracer*			gpTracer			= NULL;
DEVICE ExposureRender::Volume* 			gpVolumes			= NULL;
DEVICE ExposureRender::Light*			gpLights			= NULL;
DEVICE ExposureRender::Object*			gpObjects			= NULL;
DEVICE ExposureRender::ClippingObject*	gpClippingObjects	= NULL;
DEVICE ExposureRender::Texture*			gpTextures			= NULL;
DEVICE ExposureRender::Bitmap*			gpBitmaps			= NULL;

ExposureRender::Cuda::List<ExposureRender::Volume>						gVolumes("gpVolumes");
ExposureRender::Cuda::List<ExposureRender::Light>						gLights("gpLights");
ExposureRender::Cuda::List<ExposureRender::Object>						gObjects("gpObjects");
ExposureRender::Cuda::List<ExposureRender::ClippingObject>				gClippingObjects("gpClippingObjects");
ExposureRender::Cuda::List<ExposureRender::Texture>						gTextures("gpTextures");
ExposureRender::Cuda::List<ExposureRender::Bitmap>						gBitmaps("gpBitmaps");

ExposureRender::Cuda::SynchronizeSingle<ExposureRender::Tracer>			gTracers("gpTracer");
ExposureRender::Cuda::SynchronizeSingle<ExposureRender::FrameBuffer>	gFrameBuffers("gpFrameBuffer");

#include "utilities.h"

#include "singlescattering.cuh"
#include "estimate.cuh"
#include "toneMap.cuh"
#include "gaussianfilter.cuh"
/*
namespace ExposureRender
{

EXPOSURE_RENDER_DLL void InitializeTracer(const Tracer& Tracer)
{
	gTracers.Bind(Tracer);
	gFrameBuffers.Bind(FrameBuffer());
}

EXPOSURE_RENDER_DLL void DeinitializeTracer(int TracerID)
{
	gTracers.Unbind(TracerID);
	gFrameBuffers.Unbind(TracerID);
}

EXPOSURE_RENDER_DLL void BindTracer(const ErTracer& Tracer)
{
	gTracers.Bind(Tracer);
}

EXPOSURE_RENDER_DLL void UnbindTracer(int TracerID)
{
	gTracers.Unbind(TracerID);
}

EXPOSURE_RENDER_DLL void BindVolume(const ErVolume& Volume)
{
	gVolumes.Bind(Volume);
}

EXPOSURE_RENDER_DLL void UnbindVolume(int ID)
{
	gVolumes.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindLight(const ErLight& Light)
{
	gLights.Bind(Light);
}

EXPOSURE_RENDER_DLL void UnbindLight(int ID)
{
	gLights.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindObject(const ErObject& Object)
{
	gObjects.Bind(Object);
}

EXPOSURE_RENDER_DLL void UnbindObject(int ID)
{
	gObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindClippingObject(const ErClippingObject& ClippingObject)
{
	gClippingObjects.Bind(ClippingObject);
}

EXPOSURE_RENDER_DLL void UnbindClippingObject(int ID)
{
	gClippingObjects.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindTexture(const ErTexture& Texture)
{
	gTextures.Bind(Texture);
}

EXPOSURE_RENDER_DLL void UnbindTexture(int ID)
{
	gTextures.Unbind(ID);
}

EXPOSURE_RENDER_DLL void BindBitmap(const ErBitmap& Bitmap)
{
	gBitmaps.Bind(Bitmap);
}

EXPOSURE_RENDER_DLL void UnbindBitmap(int ID)
{
	gBitmaps.Unbind(ID);
}

EXPOSURE_RENDER_DLL void ResizeFrameBuffer(int TracerID, Resolution2i Resolution)
{
	gFrameBuffers[TracerID].Resize(Resolution);
}

EXPOSURE_RENDER_DLL void RenderEstimate(int TracerID)
{
	gTracers.Synchronize(TracerID);

	SingleScattering(gFrameBuffers[TracerID].Resolution[0], gFrameBuffers[TracerID].Resolution[1]);
	return;
	ComputeEstimate(gFrameBuffers[TracerID].Resolution[0], gFrameBuffers[TracerID].Resolution[1]);
//	FilterGaussian(Tracer.FrameBuffer.CudaFrameEstimate.GetPtr(), Tracer.FrameBuffer.CudaFrameEstimateTemp.GetPtr(), Tracer.FrameBuffer.Resolution[0], Tracer.FrameBuffer.Resolution[1]);
	ToneMap(gFrameBuffers[TracerID].Resolution[0], gFrameBuffers[TracerID].Resolution[1]);

	gTracers[TracerID].NoIterations++;
}

EXPOSURE_RENDER_DLL void GetEstimate(int TracerID, unsigned char* pData)
{
	Cuda::MemCopyDeviceToHost(gFrameBuffers[TracerID].CudaDisplayEstimate.GetPtr(), (ColorRGBAuc*)pData, gFrameBuffers[TracerID].CudaDisplayEstimate.GetNoElements());
}

EXPOSURE_RENDER_DLL void GetAutoFocusDistance(int TracerID, int FilmU, int FilmV, float& AutoFocusDistance)
{
	return;
//	ComputeAutoFocusDistance(FilmU, FilmV, AutoFocusDistance);
}

EXPOSURE_RENDER_DLL void GetNoIterations(int TracerID, int& NoIterations)
{
	NoIterations = gTracers[TracerID].NoIterations; 
}

}
*/
